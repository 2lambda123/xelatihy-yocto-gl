#include "hip/hip_runtime.h"
//
// # Yocto/CuTrace: Path tracing on Cuda/Optix
//
// Yocto/CuTrace is a simple path tracer written on the Yocto/Scene model.
// Yocto/CuTrace is implemented in `yocto_cutrace.h`, `yocto_cutrace.cpp`,
// and `yocto_cutrace.cu`.
// This library includes a stand-alone implementaton of the PCG32 random number
// generator by M.E. O'Neill.
//
// THIS IS AN EXPERIMENTAL LIBRARY THAT IS NOT READY FOR PRIME TIME
//

//
// LICENSE:
//
// Copyright (c) 2016 -- 2021 Fabio Pellacini
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//
//
// LICENSE OF INCLUDED SOFTWARE for Pcg random number generator
//
// This code also includes a small exerpt from http://www.pcg-random.org/
// licensed as follows
// *Really* minimal PCG32 code / (c) 2014 M.E. O'Neill / pcg-random.org
// Licensed under Apache License 2.0 (NO WARRANTY, etc. see website)
//

#include <optix_device.h>
// do not flip it
#include <hip/hip_runtime.h>

// HACK TO ALLOW CUT&PASTING FROM YOCTO'S CODE
#define inline __forceinline__ __device__
#define static static __forceinline__ __device__
#define optix_shader extern "C" __global__
#define optix_constant extern "C" __constant__

// whether to use builtin compound types or yocto's ones
#define CUTRACE_BUILTIN_VECS 0

// -----------------------------------------------------------------------------
// SUBSTITUTES FOR STD TYPES
// -----------------------------------------------------------------------------
namespace yocto {

// pair
template <typename T1, typename T2>
struct pair {
  T1 first;
  T2 second;
};

template <typename T>
struct span {
  inline bool   empty() const { return _size == 0; }
  inline size_t size() const { return _size; }

  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }
  inline T&       at(int idx) { return _data[idx]; }
  inline const T& at(int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline T*       data() { return _data; }
  inline const T* data() const { return _data; }

  T*     _data = nullptr;
  size_t _size = 0;
};

}  // namespace yocto

// -----------------------------------------------------------------------------
// MATH TYPES
// -----------------------------------------------------------------------------
namespace yocto {

#if CUTRACE_BUILTIN_VECS

using vec2f = float2;
using vec3f = float3;
using vec4f = float4;
using vec2i = int2;
using vec3i = int3;
using vec4i = int4;

#else

struct vec2f {
  float x = 0;
  float y = 0;

  inline float&       operator[](int idx) { return (&x)[idx]; }
  inline const float& operator[](int idx) const { return (&x)[idx]; }
};

struct vec3f {
  float x = 0;
  float y = 0;
  float z = 0;

  inline float&       operator[](int idx) { return (&x)[idx]; }
  inline const float& operator[](int idx) const { return (&x)[idx]; }
};

struct vec4f {
  float x = 0;
  float y = 0;
  float z = 0;
  float w = 0;

  inline float&       operator[](int idx) { return (&x)[idx]; }
  inline const float& operator[](int idx) const { return (&x)[idx]; }
};

struct vec2i {
  int x = 0;
  int y = 0;
};

struct vec3i {
  int x = 0;
  int y = 0;
  int z = 0;
};

struct vec4i {
  int x = 0;
  int y = 0;
  int z = 0;
  int w = 0;
};

using byte = unsigned char;

struct vec4b {
  byte x = 0;
  byte y = 0;
  byte z = 0;
  byte w = 0;
};

#endif

// Rigid frames stored as a column-major affine transform matrix.
struct frame2f {
  vec2f x = {1, 0};
  vec2f y = {0, 1};
  vec2f o = {0, 0};
};

// Rigid frames stored as a column-major affine transform matrix.
struct frame3f {
  vec3f x = {1, 0, 0};
  vec3f y = {0, 1, 0};
  vec3f z = {0, 0, 1};
  vec3f o = {0, 0, 0};
};

}  // namespace yocto

// -----------------------------------------------------------------------------
// MATH CONSTANTS AND FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

using byte   = unsigned char;
using uint   = unsigned int;
using ushort = unsigned short;

constexpr float flt_max = 1e20f;  // TODO: numerical limits

constexpr double pi  = 3.14159265358979323846;
constexpr float  pif = (float)pi;

inline float abs(float a) { return a < 0 ? -a : a; }
inline float min(float a, float b) { return (a < b) ? a : b; }
inline float max(float a, float b) { return (a > b) ? a : b; }
inline float clamp(float a, float min_, float max_) {
  return min(max(a, min_), max_);
}
inline float sign(float a) { return a < 0 ? -1.0f : 1.0f; }
inline float sqr(float a) { return a * a; }
inline float sqrt(float a) { return std::sqrt(a); }
inline float sin(float a) { return std::sin(a); }
inline float cos(float a) { return std::cos(a); }
inline float tan(float a) { return std::tan(a); }
inline float asin(float a) { return std::asin(a); }
inline float acos(float a) { return std::acos(a); }
inline float atan(float a) { return std::atan(a); }
inline float log(float a) { return std::log(a); }
inline float exp(float a) { return std::exp(a); }
inline float log2(float a) { return std::log2(a); }
inline float exp2(float a) { return std::exp2(a); }
inline float pow(float a, float b) { return std::pow(a, b); }
inline bool  isfinite(float a) { return ::isfinite(a); }
inline float atan2(float a, float b) { return std::atan2(a, b); }
inline float fmod(float a, float b) { return std::fmod(a, b); }
inline void  swap(float& a, float& b) { std::swap(a, b); }
inline float radians(float a) { return a * pif / 180; }
inline float degrees(float a) { return a * 180 / pif; }
inline float lerp(float a, float b, float u) { return a * (1 - u) + b * u; }
inline float step(float a, float u) { return u < a ? 0.0f : 1.0f; }
inline float smoothstep(float a, float b, float u) {
  auto t = clamp((u - a) / (b - a), 0.0f, 1.0f);
  return t * t * (3 - 2 * t);
}
inline float bias(float a, float bias) {
  return a / ((1 / bias - 2) * (1 - a) + 1);
}
inline float gain(float a, float gain) {
  return (a < 0.5f) ? bias(a * 2, gain) / 2
                    : bias(a * 2 - 1, 1 - gain) / 2 + 0.5f;
}

inline int  abs(int a) { return a < 0 ? -a : a; }
inline int  min(int a, int b) { return (a < b) ? a : b; }
inline int  max(int a, int b) { return (a > b) ? a : b; }
inline int  clamp(int a, int min_, int max_) { return min(max(a, min_), max_); }
inline int  sign(int a) { return a < 0 ? -1 : 1; }
inline int  pow2(int a) { return 1 << a; }
inline void swap(int& a, int& b) { std::swap(a, b); }

inline size_t min(size_t a, size_t b) { return (a < b) ? a : b; }
inline size_t max(size_t a, size_t b) { return (a > b) ? a : b; }

}  // namespace yocto

// -----------------------------------------------------------------------------
// VECTORS
// -----------------------------------------------------------------------------
namespace yocto {

// Element access
inline vec3f xyz(const vec4f& a) { return {a.x, a.y, a.z}; }

// Vector sequence operations.
inline int          size(const vec2f& a) { return 2; }
inline const float* begin(const vec2f& a) { return &a.x; }
inline const float* end(const vec2f& a) { return &a.x + 2; }
inline float*       begin(vec2f& a) { return &a.x; }
inline float*       end(vec2f& a) { return &a.x + 2; }
inline const float* data(const vec2f& a) { return &a.x; }
inline float*       data(vec2f& a) { return &a.x; }

// Vector comparison operations.
inline bool operator==(const vec2f& a, const vec2f& b) {
  return a.x == b.x && a.y == b.y;
}
inline bool operator!=(const vec2f& a, const vec2f& b) {
  return a.x != b.x || a.y != b.y;
}

// Vector operations.
inline vec2f operator+(const vec2f& a) { return a; }
inline vec2f operator-(const vec2f& a) { return {-a.x, -a.y}; }
inline vec2f operator+(const vec2f& a, const vec2f& b) {
  return {a.x + b.x, a.y + b.y};
}
inline vec2f operator+(const vec2f& a, float b) { return {a.x + b, a.y + b}; }
inline vec2f operator+(float a, const vec2f& b) { return {a + b.x, a + b.y}; }
inline vec2f operator-(const vec2f& a, const vec2f& b) {
  return {a.x - b.x, a.y - b.y};
}
inline vec2f operator-(const vec2f& a, float b) { return {a.x - b, a.y - b}; }
inline vec2f operator-(float a, const vec2f& b) { return {a - b.x, a - b.y}; }
inline vec2f operator*(const vec2f& a, const vec2f& b) {
  return {a.x * b.x, a.y * b.y};
}
inline vec2f operator*(const vec2f& a, float b) { return {a.x * b, a.y * b}; }
inline vec2f operator*(float a, const vec2f& b) { return {a * b.x, a * b.y}; }
inline vec2f operator/(const vec2f& a, const vec2f& b) {
  return {a.x / b.x, a.y / b.y};
}
inline vec2f operator/(const vec2f& a, float b) { return {a.x / b, a.y / b}; }
inline vec2f operator/(float a, const vec2f& b) { return {a / b.x, a / b.y}; }

// Vector assignments
inline vec2f& operator+=(vec2f& a, const vec2f& b) { return a = a + b; }
inline vec2f& operator+=(vec2f& a, float b) { return a = a + b; }
inline vec2f& operator-=(vec2f& a, const vec2f& b) { return a = a - b; }
inline vec2f& operator-=(vec2f& a, float b) { return a = a - b; }
inline vec2f& operator*=(vec2f& a, const vec2f& b) { return a = a * b; }
inline vec2f& operator*=(vec2f& a, float b) { return a = a * b; }
inline vec2f& operator/=(vec2f& a, const vec2f& b) { return a = a / b; }
inline vec2f& operator/=(vec2f& a, float b) { return a = a / b; }

// Vector products and lengths.
inline float dot(const vec2f& a, const vec2f& b) {
  return a.x * b.x + a.y * b.y;
}
inline float cross(const vec2f& a, const vec2f& b) {
  return a.x * b.y - a.y * b.x;
}

inline float length(const vec2f& a) { return sqrt(dot(a, a)); }
inline float length_squared(const vec2f& a) { return dot(a, a); }
inline vec2f normalize(const vec2f& a) {
  auto l = length(a);
  return (l != 0) ? a / l : a;
}
inline float distance(const vec2f& a, const vec2f& b) { return length(a - b); }
inline float distance_squared(const vec2f& a, const vec2f& b) {
  return dot(a - b, a - b);
}
inline float angle(const vec2f& a, const vec2f& b) {
  return acos(clamp(dot(normalize(a), normalize(b)), (float)-1, (float)1));
}

// Max element and clamp.
inline vec2f max(const vec2f& a, float b) { return {max(a.x, b), max(a.y, b)}; }
inline vec2f min(const vec2f& a, float b) { return {min(a.x, b), min(a.y, b)}; }
inline vec2f max(const vec2f& a, const vec2f& b) {
  return {max(a.x, b.x), max(a.y, b.y)};
}
inline vec2f min(const vec2f& a, const vec2f& b) {
  return {min(a.x, b.x), min(a.y, b.y)};
}
inline vec2f clamp(const vec2f& x, float min, float max) {
  return {clamp(x.x, min, max), clamp(x.y, min, max)};
}
inline vec2f lerp(const vec2f& a, const vec2f& b, float u) {
  return a * (1 - u) + b * u;
}
inline vec2f lerp(const vec2f& a, const vec2f& b, const vec2f& u) {
  return a * (1 - u) + b * u;
}

inline float max(const vec2f& a) { return max(a.x, a.y); }
inline float min(const vec2f& a) { return min(a.x, a.y); }
inline float sum(const vec2f& a) { return a.x + a.y; }
inline float mean(const vec2f& a) { return sum(a) / 2; }

// Functions applied to vector elements
inline vec2f abs(const vec2f& a) { return {abs(a.x), abs(a.y)}; }
inline vec2f sqr(const vec2f& a) { return {sqr(a.x), sqr(a.y)}; }
inline vec2f sqrt(const vec2f& a) { return {sqrt(a.x), sqrt(a.y)}; }
inline vec2f exp(const vec2f& a) { return {exp(a.x), exp(a.y)}; }
inline vec2f log(const vec2f& a) { return {log(a.x), log(a.y)}; }
inline vec2f exp2(const vec2f& a) { return {exp2(a.x), exp2(a.y)}; }
inline vec2f log2(const vec2f& a) { return {log2(a.x), log2(a.y)}; }
inline bool  isfinite(const vec2f& a) { return isfinite(a.x) && isfinite(a.y); }
inline vec2f pow(const vec2f& a, float b) { return {pow(a.x, b), pow(a.y, b)}; }
inline vec2f pow(const vec2f& a, const vec2f& b) {
  return {pow(a.x, b.x), pow(a.y, b.y)};
}
inline vec2f gain(const vec2f& a, float b) {
  return {gain(a.x, b), gain(a.y, b)};
}
inline void swap(vec2f& a, vec2f& b) { std::swap(a, b); }

// Vector sequence operations.
inline int          size(const vec3f& a) { return 3; }
inline const float* begin(const vec3f& a) { return &a.x; }
inline const float* end(const vec3f& a) { return &a.x + 3; }
inline float*       begin(vec3f& a) { return &a.x; }
inline float*       end(vec3f& a) { return &a.x + 3; }
inline const float* data(const vec3f& a) { return &a.x; }
inline float*       data(vec3f& a) { return &a.x; }

// Vector comparison operations.
inline bool operator==(const vec3f& a, const vec3f& b) {
  return a.x == b.x && a.y == b.y && a.z == b.z;
}
inline bool operator!=(const vec3f& a, const vec3f& b) {
  return a.x != b.x || a.y != b.y || a.z != b.z;
}

// Vector operations.
inline vec3f operator+(const vec3f& a) { return a; }
inline vec3f operator-(const vec3f& a) { return {-a.x, -a.y, -a.z}; }
inline vec3f operator+(const vec3f& a, const vec3f& b) {
  return {a.x + b.x, a.y + b.y, a.z + b.z};
}
inline vec3f operator+(const vec3f& a, float b) {
  return {a.x + b, a.y + b, a.z + b};
}
inline vec3f operator+(float a, const vec3f& b) {
  return {a + b.x, a + b.y, a + b.z};
}
inline vec3f operator-(const vec3f& a, const vec3f& b) {
  return {a.x - b.x, a.y - b.y, a.z - b.z};
}
inline vec3f operator-(const vec3f& a, float b) {
  return {a.x - b, a.y - b, a.z - b};
}
inline vec3f operator-(float a, const vec3f& b) {
  return {a - b.x, a - b.y, a - b.z};
}
inline vec3f operator*(const vec3f& a, const vec3f& b) {
  return {a.x * b.x, a.y * b.y, a.z * b.z};
}
inline vec3f operator*(const vec3f& a, float b) {
  return {a.x * b, a.y * b, a.z * b};
}
inline vec3f operator*(float a, const vec3f& b) {
  return {a * b.x, a * b.y, a * b.z};
}
inline vec3f operator/(const vec3f& a, const vec3f& b) {
  return {a.x / b.x, a.y / b.y, a.z / b.z};
}
inline vec3f operator/(const vec3f& a, float b) {
  return {a.x / b, a.y / b, a.z / b};
}
inline vec3f operator/(float a, const vec3f& b) {
  return {a / b.x, a / b.y, a / b.z};
}

// Vector assignments
inline vec3f& operator+=(vec3f& a, const vec3f& b) { return a = a + b; }
inline vec3f& operator+=(vec3f& a, float b) { return a = a + b; }
inline vec3f& operator-=(vec3f& a, const vec3f& b) { return a = a - b; }
inline vec3f& operator-=(vec3f& a, float b) { return a = a - b; }
inline vec3f& operator*=(vec3f& a, const vec3f& b) { return a = a * b; }
inline vec3f& operator*=(vec3f& a, float b) { return a = a * b; }
inline vec3f& operator/=(vec3f& a, const vec3f& b) { return a = a / b; }
inline vec3f& operator/=(vec3f& a, float b) { return a = a / b; }

// Vector products and lengths.
inline float dot(const vec3f& a, const vec3f& b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}
inline vec3f cross(const vec3f& a, const vec3f& b) {
  return {a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x};
}

inline float length(const vec3f& a) { return sqrt(dot(a, a)); }
inline float length_squared(const vec3f& a) { return dot(a, a); }
inline vec3f normalize(const vec3f& a) {
  auto l = length(a);
  return (l != 0) ? a / l : a;
}
inline float distance(const vec3f& a, const vec3f& b) { return length(a - b); }
inline float distance_squared(const vec3f& a, const vec3f& b) {
  return dot(a - b, a - b);
}
inline float angle(const vec3f& a, const vec3f& b) {
  return acos(clamp(dot(normalize(a), normalize(b)), (float)-1, (float)1));
}

// Orthogonal vectors.
inline vec3f orthogonal(const vec3f& v) {
  // http://lolengine.net/blog/2013/09/21/picking-orthogonal-vector-combing-coconuts)
  return abs(v.x) > abs(v.z) ? vec3f{-v.y, v.x, 0} : vec3f{0, -v.z, v.y};
}
inline vec3f orthonormalize(const vec3f& a, const vec3f& b) {
  return normalize(a - b * dot(a, b));
}

// Reflected and refracted vector.
inline vec3f reflect(const vec3f& w, const vec3f& n) {
  return -w + 2 * dot(n, w) * n;
}
inline vec3f refract(const vec3f& w, const vec3f& n, float inv_eta) {
  auto cosine = dot(n, w);
  auto k      = 1 + inv_eta * inv_eta * (cosine * cosine - 1);
  if (k < 0) return {0, 0, 0};  // tir
  return -w * inv_eta + (inv_eta * cosine - sqrt(k)) * n;
}

// Max element and clamp.
inline vec3f max(const vec3f& a, float b) {
  return {max(a.x, b), max(a.y, b), max(a.z, b)};
}
inline vec3f min(const vec3f& a, float b) {
  return {min(a.x, b), min(a.y, b), min(a.z, b)};
}
inline vec3f max(const vec3f& a, const vec3f& b) {
  return {max(a.x, b.x), max(a.y, b.y), max(a.z, b.z)};
}
inline vec3f min(const vec3f& a, const vec3f& b) {
  return {min(a.x, b.x), min(a.y, b.y), min(a.z, b.z)};
}
inline vec3f clamp(const vec3f& x, float min, float max) {
  return {clamp(x.x, min, max), clamp(x.y, min, max), clamp(x.z, min, max)};
}
inline vec3f lerp(const vec3f& a, const vec3f& b, float u) {
  return a * (1 - u) + b * u;
}
inline vec3f lerp(const vec3f& a, const vec3f& b, const vec3f& u) {
  return a * (1 - u) + b * u;
}

inline float max(const vec3f& a) { return max(max(a.x, a.y), a.z); }
inline float min(const vec3f& a) { return min(min(a.x, a.y), a.z); }
inline float sum(const vec3f& a) { return a.x + a.y + a.z; }
inline float mean(const vec3f& a) { return sum(a) / 3; }

// Functions applied to vector elements
inline vec3f abs(const vec3f& a) { return {abs(a.x), abs(a.y), abs(a.z)}; }
inline vec3f sqr(const vec3f& a) { return {sqr(a.x), sqr(a.y), sqr(a.z)}; }
inline vec3f sqrt(const vec3f& a) { return {sqrt(a.x), sqrt(a.y), sqrt(a.z)}; }
inline vec3f exp(const vec3f& a) { return {exp(a.x), exp(a.y), exp(a.z)}; }
inline vec3f log(const vec3f& a) { return {log(a.x), log(a.y), log(a.z)}; }
inline vec3f exp2(const vec3f& a) { return {exp2(a.x), exp2(a.y), exp2(a.z)}; }
inline vec3f log2(const vec3f& a) { return {log2(a.x), log2(a.y), log2(a.z)}; }
inline vec3f pow(const vec3f& a, float b) {
  return {pow(a.x, b), pow(a.y, b), pow(a.z, b)};
}
inline vec3f pow(const vec3f& a, const vec3f& b) {
  return {pow(a.x, b.x), pow(a.y, b.y), pow(a.z, b.z)};
}
inline vec3f gain(const vec3f& a, float b) {
  return {gain(a.x, b), gain(a.y, b), gain(a.z, b)};
}
inline bool isfinite(const vec3f& a) {
  return isfinite(a.x) && isfinite(a.y) && isfinite(a.z);
}
inline void swap(vec3f& a, vec3f& b) { std::swap(a, b); }

// Vector sequence operations.
inline int          size(const vec4f& a) { return 4; }
inline const float* begin(const vec4f& a) { return &a.x; }
inline const float* end(const vec4f& a) { return &a.x + 4; }
inline float*       begin(vec4f& a) { return &a.x; }
inline float*       end(vec4f& a) { return &a.x + 4; }
inline const float* data(const vec4f& a) { return &a.x; }
inline float*       data(vec4f& a) { return &a.x; }

// Vector comparison operations.
inline bool operator==(const vec4f& a, const vec4f& b) {
  return a.x == b.x && a.y == b.y && a.z == b.z && a.w == b.w;
}
inline bool operator!=(const vec4f& a, const vec4f& b) {
  return a.x != b.x || a.y != b.y || a.z != b.z || a.w != b.w;
}

// Vector operations.
inline vec4f operator+(const vec4f& a) { return a; }
inline vec4f operator-(const vec4f& a) { return {-a.x, -a.y, -a.z, -a.w}; }
inline vec4f operator+(const vec4f& a, const vec4f& b) {
  return {a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w};
}
inline vec4f operator+(const vec4f& a, float b) {
  return {a.x + b, a.y + b, a.z + b, a.w + b};
}
inline vec4f operator+(float a, const vec4f& b) {
  return {a + b.x, a + b.y, a + b.z, a + b.w};
}
inline vec4f operator-(const vec4f& a, const vec4f& b) {
  return {a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w};
}
inline vec4f operator-(const vec4f& a, float b) {
  return {a.x - b, a.y - b, a.z - b, a.w - b};
}
inline vec4f operator-(float a, const vec4f& b) {
  return {a - b.x, a - b.y, a - b.z, a - b.w};
}
inline vec4f operator*(const vec4f& a, const vec4f& b) {
  return {a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w};
}
inline vec4f operator*(const vec4f& a, float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}
inline vec4f operator*(float a, const vec4f& b) {
  return {a * b.x, a * b.y, a * b.z, a * b.w};
}
inline vec4f operator/(const vec4f& a, const vec4f& b) {
  return {a.x / b.x, a.y / b.y, a.z / b.z, a.w / b.w};
}
inline vec4f operator/(const vec4f& a, float b) {
  return {a.x / b, a.y / b, a.z / b, a.w / b};
}
inline vec4f operator/(float a, const vec4f& b) {
  return {a / b.x, a / b.y, a / b.z, a / b.w};
}

// Vector assignments
inline vec4f& operator+=(vec4f& a, const vec4f& b) { return a = a + b; }
inline vec4f& operator+=(vec4f& a, float b) { return a = a + b; }
inline vec4f& operator-=(vec4f& a, const vec4f& b) { return a = a - b; }
inline vec4f& operator-=(vec4f& a, float b) { return a = a - b; }
inline vec4f& operator*=(vec4f& a, const vec4f& b) { return a = a * b; }
inline vec4f& operator*=(vec4f& a, float b) { return a = a * b; }
inline vec4f& operator/=(vec4f& a, const vec4f& b) { return a = a / b; }
inline vec4f& operator/=(vec4f& a, float b) { return a = a / b; }

// Vector products and lengths.
inline float dot(const vec4f& a, const vec4f& b) {
  return a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
}
inline float length(const vec4f& a) { return sqrt(dot(a, a)); }
inline float length_squared(const vec4f& a) { return dot(a, a); }
inline vec4f normalize(const vec4f& a) {
  auto l = length(a);
  return (l != 0) ? a / l : a;
}
inline float distance(const vec4f& a, const vec4f& b) { return length(a - b); }
inline float distance_squared(const vec4f& a, const vec4f& b) {
  return dot(a - b, a - b);
}
inline float angle(const vec4f& a, const vec4f& b) {
  return acos(clamp(dot(normalize(a), normalize(b)), (float)-1, (float)1));
}

inline vec4f slerp(const vec4f& a, const vec4f& b, float u) {
  // https://en.wikipedia.org/wiki/Slerp
  auto an = normalize(a), bn = normalize(b);
  auto d = dot(an, bn);
  if (d < 0) {
    bn = -bn;
    d  = -d;
  }
  if (d > (float)0.9995) return normalize(an + u * (bn - an));
  auto th = acos(clamp(d, (float)-1, (float)1));
  if (th == 0) return an;
  return an * (sin(th * (1 - u)) / sin(th)) + bn * (sin(th * u) / sin(th));
}

// Max element and clamp.
inline vec4f max(const vec4f& a, float b) {
  return {max(a.x, b), max(a.y, b), max(a.z, b), max(a.w, b)};
}
inline vec4f min(const vec4f& a, float b) {
  return {min(a.x, b), min(a.y, b), min(a.z, b), min(a.w, b)};
}
inline vec4f max(const vec4f& a, const vec4f& b) {
  return {max(a.x, b.x), max(a.y, b.y), max(a.z, b.z), max(a.w, b.w)};
}
inline vec4f min(const vec4f& a, const vec4f& b) {
  return {min(a.x, b.x), min(a.y, b.y), min(a.z, b.z), min(a.w, b.w)};
}
inline vec4f clamp(const vec4f& x, float min, float max) {
  return {clamp(x.x, min, max), clamp(x.y, min, max), clamp(x.z, min, max),
      clamp(x.w, min, max)};
}
inline vec4f lerp(const vec4f& a, const vec4f& b, float u) {
  return a * (1 - u) + b * u;
}
inline vec4f lerp(const vec4f& a, const vec4f& b, const vec4f& u) {
  return a * (1 - u) + b * u;
}

inline float max(const vec4f& a) { return max(max(max(a.x, a.y), a.z), a.w); }
inline float min(const vec4f& a) { return min(min(min(a.x, a.y), a.z), a.w); }
inline float sum(const vec4f& a) { return a.x + a.y + a.z + a.w; }
inline float mean(const vec4f& a) { return sum(a) / 4; }

// Functions applied to vector elements
inline vec4f abs(const vec4f& a) {
  return {abs(a.x), abs(a.y), abs(a.z), abs(a.w)};
}
inline vec4f sqr(const vec4f& a) {
  return {sqr(a.x), sqr(a.y), sqr(a.z), sqr(a.w)};
}
inline vec4f sqrt(const vec4f& a) {
  return {sqrt(a.x), sqrt(a.y), sqrt(a.z), sqrt(a.w)};
}
inline vec4f exp(const vec4f& a) {
  return {exp(a.x), exp(a.y), exp(a.z), exp(a.w)};
}
inline vec4f log(const vec4f& a) {
  return {log(a.x), log(a.y), log(a.z), log(a.w)};
}
inline vec4f exp2(const vec4f& a) {
  return {exp2(a.x), exp2(a.y), exp2(a.z), exp2(a.w)};
}
inline vec4f log2(const vec4f& a) {
  return {log2(a.x), log2(a.y), log2(a.z), log2(a.w)};
}
inline vec4f pow(const vec4f& a, float b) {
  return {pow(a.x, b), pow(a.y, b), pow(a.z, b), pow(a.w, b)};
}
inline vec4f pow(const vec4f& a, const vec4f& b) {
  return {pow(a.x, b.x), pow(a.y, b.y), pow(a.z, b.z), pow(a.w, b.w)};
}
inline vec4f gain(const vec4f& a, float b) {
  return {gain(a.x, b), gain(a.y, b), gain(a.z, b), gain(a.w, b)};
}
inline bool isfinite(const vec4f& a) {
  return isfinite(a.x) && isfinite(a.y) && isfinite(a.z) && isfinite(a.w);
}
inline void swap(vec4f& a, vec4f& b) { std::swap(a, b); }

// Quaternion operatons represented as xi + yj + zk + w
// const auto identity_quat4f = vec4f{0, 0, 0, 1};
inline vec4f quat_mul(const vec4f& a, float b) {
  return {a.x * b, a.y * b, a.z * b, a.w * b};
}
inline vec4f quat_mul(const vec4f& a, const vec4f& b) {
  return {a.x * b.w + a.w * b.x + a.y * b.w - a.z * b.y,
      a.y * b.w + a.w * b.y + a.z * b.x - a.x * b.z,
      a.z * b.w + a.w * b.z + a.x * b.y - a.y * b.x,
      a.w * b.w - a.x * b.x - a.y * b.y - a.z * b.z};
}
inline vec4f quat_conjugate(const vec4f& a) { return {-a.x, -a.y, -a.z, a.w}; }
inline vec4f quat_inverse(const vec4f& a) {
  return quat_conjugate(a) / dot(a, a);
}

// Frame construction from axis.
inline frame3f frame_fromz(const vec3f& o, const vec3f& v) {
  // https://graphics.pixar.com/library/OrthonormalB/paper.pdf
  auto z    = normalize(v);
  auto sign = copysignf(1.0f, z.z);
  auto a    = -1.0f / (sign + z.z);
  auto b    = z.x * z.y * a;
  auto x    = vec3f{1.0f + sign * z.x * z.x * a, sign * b, -sign * z.x};
  auto y    = vec3f{b, sign + z.y * z.y * a, -z.y};
  return {x, y, z, o};
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRANSFORMS
// -----------------------------------------------------------------------------
namespace yocto {

// Transforms points, vectors and directions by frames.
inline vec2f transform_point(const frame2f& a, const vec2f& b) {
  return a.x * b.x + a.y * b.y + a.o;
}
inline vec2f transform_vector(const frame2f& a, const vec2f& b) {
  return a.x * b.x + a.y * b.y;
}
inline vec2f transform_direction(const frame2f& a, const vec2f& b) {
  return normalize(transform_vector(a, b));
}
inline vec2f transform_normal(
    const frame2f& a, const vec2f& b, bool non_rigid) {
  // if (non_rigid) {
  //  return transform_normal(rotation(a), b);
  //} else {
  return normalize(transform_vector(a, b));
  //}
}

// Transforms points, vectors and directions by frames.
inline vec3f transform_point(const frame3f& a, const vec3f& b) {
  return a.x * b.x + a.y * b.y + a.z * b.z + a.o;
}
inline vec3f transform_vector(const frame3f& a, const vec3f& b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}
inline vec3f transform_direction(const frame3f& a, const vec3f& b) {
  return normalize(transform_vector(a, b));
}
inline vec3f transform_normal(
    const frame3f& a, const vec3f& b, bool non_rigid = false) {
  // if (non_rigid) {
  //   return transform_normal(rotation(a), b);
  // } else {
  return normalize(transform_vector(a, b));
  //}
}

// Translation, scaling and rotations transforms.
inline frame3f translation_frame(const vec3f& a) {
  return {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, a};
}
inline frame3f scaling_frame(const vec3f& a) {
  return {{a.x, 0, 0}, {0, a.y, 0}, {0, 0, a.z}, {0, 0, 0}};
}
inline frame3f rotation_frame(const vec3f& axis, float angle) {
  auto s = sin(angle), c = cos(angle);
  auto vv = normalize(axis);
  return {{c + (1 - c) * vv.x * vv.x, (1 - c) * vv.x * vv.y + s * vv.z,
              (1 - c) * vv.x * vv.z - s * vv.y},
      {(1 - c) * vv.x * vv.y - s * vv.z, c + (1 - c) * vv.y * vv.y,
          (1 - c) * vv.y * vv.z + s * vv.x},
      {(1 - c) * vv.x * vv.z + s * vv.y, (1 - c) * vv.y * vv.z - s * vv.x,
          c + (1 - c) * vv.z * vv.z},
      {0, 0, 0}};
}

// Lookat frame. Z-axis can be inverted with inv_xz.
inline frame3f lookat_frame(
    const vec3f& eye, const vec3f& center, const vec3f& up, bool inv_xz) {
  auto w = normalize(eye - center);
  auto u = normalize(cross(up, w));
  auto v = normalize(cross(w, u));
  if (inv_xz) {
    w = -w;
    u = -u;
  }
  return {u, v, w, eye};
}

// Additions
inline vec3f transform_point_inverse(const frame3f& frame, const vec3f& v) {
  return {dot(frame.x, v - frame.o), dot(frame.y, v - frame.o),
      dot(frame.z, v - frame.o)};
}
inline vec3f transform_vector_inverse(const frame3f& frame, const vec3f& v) {
  return {dot(frame.x, v), dot(frame.y, v), dot(frame.z, v)};
}
inline vec3f transform_direction_inverse(const frame3f& frame, const vec3f& v) {
  return normalize(vec3f{dot(frame.x, v), dot(frame.y, v), dot(frame.z, v)});
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// COLOR
// -----------------------------------------------------------------------------
namespace yocto {

// sRGB non-linear curve
inline float srgb_to_rgb(float srgb) {
  return (srgb <= 0.04045) ? srgb / 12.92f
                           : pow((srgb + 0.055f) / (1.0f + 0.055f), 2.4f);
}
inline float rgb_to_srgb(float rgb) {
  return (rgb <= 0.0031308f) ? 12.92f * rgb
                             : (1 + 0.055f) * pow(rgb, 1 / 2.4f) - 0.055f;
}
inline vec3f srgb_to_rgb(const vec3f& srgb) {
  return {srgb_to_rgb(srgb.x), srgb_to_rgb(srgb.y), srgb_to_rgb(srgb.z)};
}
inline vec4f srgb_to_rgb(const vec4f& srgb) {
  return {
      srgb_to_rgb(srgb.x), srgb_to_rgb(srgb.y), srgb_to_rgb(srgb.z), srgb.w};
}
inline vec3f rgb_to_srgb(const vec3f& rgb) {
  return {rgb_to_srgb(rgb.x), rgb_to_srgb(rgb.y), rgb_to_srgb(rgb.z)};
}
inline vec4f rgb_to_srgb(const vec4f& rgb) {
  return {rgb_to_srgb(rgb.x), rgb_to_srgb(rgb.y), rgb_to_srgb(rgb.z), rgb.w};
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// GEOMETRY TYPES
// -----------------------------------------------------------------------------
namespace yocto {

// Ray epsilon
constexpr auto ray_eps = 1e-4f;
constexpr auto ray_far = 1e20f;

struct ray2f {
  vec2f o    = {0, 0};
  vec2f d    = {0, 1};
  float tmin = ray_eps;
  float tmax = ray_far;
};

// Rays with origin, direction and min/max t value.
struct ray3f {
  vec3f o    = {0, 0, 0};
  vec3f d    = {0, 0, 1};
  float tmin = ray_eps;
  float tmax = ray_far;
};

}  // namespace yocto

// -----------------------------------------------------------------------------
// GEOMETRY UTILITIES
// -----------------------------------------------------------------------------
namespace yocto {

// Line properties.
inline vec3f line_tangent(const vec3f& p0, const vec3f& p1) {
  return normalize(p1 - p0);
}
inline float line_length(const vec3f& p0, const vec3f& p1) {
  return length(p1 - p0);
}

// Triangle properties.
inline vec3f triangle_normal(
    const vec3f& p0, const vec3f& p1, const vec3f& p2) {
  return normalize(cross(p1 - p0, p2 - p0));
}
inline float triangle_area(const vec3f& p0, const vec3f& p1, const vec3f& p2) {
  return length(cross(p1 - p0, p2 - p0)) / 2;
}

// Quad propeties.
inline vec3f quad_normal(
    const vec3f& p0, const vec3f& p1, const vec3f& p2, const vec3f& p3) {
  return normalize(triangle_normal(p0, p1, p3) + triangle_normal(p2, p3, p1));
}
inline float quad_area(
    const vec3f& p0, const vec3f& p1, const vec3f& p2, const vec3f& p3) {
  return triangle_area(p0, p1, p3) + triangle_area(p2, p3, p1);
}

// Interpolates values over a line parameterized from a to b by u. Same as lerp.
template <typename T>
inline T interpolate_line(const T& p0, const T& p1, float u) {
  return p0 * (1 - u) + p1 * u;
}
// Interpolates values over a triangle parameterized by u and v along the
// (p1-p0) and (p2-p0) directions. Same as barycentric interpolation.
template <typename T>
inline T interpolate_triangle(
    const T& p0, const T& p1, const T& p2, const vec2f& uv) {
  return p0 * (1 - uv.x - uv.y) + p1 * uv.x + p2 * uv.y;
}
// Interpolates values over a quad parameterized by u and v along the
// (p1-p0) and (p2-p1) directions. Same as bilinear interpolation.
template <typename T>
inline T interpolate_quad(
    const T& p0, const T& p1, const T& p2, const T& p3, const vec2f& uv) {
  if (uv.x + uv.y <= 1) {
    return interpolate_triangle(p0, p1, p3, uv);
  } else {
    return interpolate_triangle(p2, p3, p1, 1 - uv);
  }
}

// Interpolates values along a cubic Bezier segment parametrized by u.
template <typename T>
inline T interpolate_bezier(
    const T& p0, const T& p1, const T& p2, const T& p3, float u) {
  return p0 * (1 - u) * (1 - u) * (1 - u) + p1 * 3 * u * (1 - u) * (1 - u) +
         p2 * 3 * u * u * (1 - u) + p3 * u * u * u;
}
// Computes the derivative of a cubic Bezier segment parametrized by u.
template <typename T>
inline T interpolate_bezier_derivative(
    const T& p0, const T& p1, const T& p2, const T& p3, float u) {
  return (p1 - p0) * 3 * (1 - u) * (1 - u) + (p2 - p1) * 6 * u * (1 - u) +
         (p3 - p2) * 3 * u * u;
}

// Interpolated line properties.
inline vec3f line_point(const vec3f& p0, const vec3f& p1, float u) {
  return p0 * (1 - u) + p1 * u;
}
inline vec3f line_tangent(const vec3f& t0, const vec3f& t1, float u) {
  return normalize(t0 * (1 - u) + t1 * u);
}

// Interpolated triangle properties.
inline vec3f triangle_point(
    const vec3f& p0, const vec3f& p1, const vec3f& p2, const vec2f& uv) {
  return p0 * (1 - uv.x - uv.y) + p1 * uv.x + p2 * uv.y;
}
inline vec3f triangle_normal(
    const vec3f& n0, const vec3f& n1, const vec3f& n2, const vec2f& uv) {
  return normalize(n0 * (1 - uv.x - uv.y) + n1 * uv.x + n2 * uv.y);
}

// Interpolated quad properties.
inline vec3f quad_point(const vec3f& p0, const vec3f& p1, const vec3f& p2,
    const vec3f& p3, const vec2f& uv) {
  if (uv.x + uv.y <= 1) {
    return triangle_point(p0, p1, p3, uv);
  } else {
    return triangle_point(p2, p3, p1, 1 - uv);
  }
}
inline vec3f quad_normal(const vec3f& n0, const vec3f& n1, const vec3f& n2,
    const vec3f& n3, const vec2f& uv) {
  if (uv.x + uv.y <= 1) {
    return triangle_normal(n0, n1, n3, uv);
  } else {
    return triangle_normal(n2, n3, n1, 1 - uv);
  }
}

// Interpolated sphere properties.
inline vec3f sphere_point(const vec3f p, float r, const vec2f& uv) {
  return p + r * vec3f{cos(uv.x * 2 * pif) * sin(uv.y * pif),
                     sin(uv.x * 2 * pif) * sin(uv.y * pif), cos(uv.y * pif)};
}
inline vec3f sphere_normal(const vec3f p, float r, const vec2f& uv) {
  return normalize(vec3f{cos(uv.x * 2 * pif) * sin(uv.y * pif),
      sin(uv.x * 2 * pif) * sin(uv.y * pif), cos(uv.y * pif)});
}

// Triangle tangent and bitangent from uv
inline pair<vec3f, vec3f> triangle_tangents_fromuv(const vec3f& p0,
    const vec3f& p1, const vec3f& p2, const vec2f& uv0, const vec2f& uv1,
    const vec2f& uv2) {
  // Follows the definition in http://www.terathon.com/code/tangent.html and
  // https://gist.github.com/aras-p/2843984
  // normal points up from texture space
  auto p   = p1 - p0;
  auto q   = p2 - p0;
  auto s   = vec2f{uv1.x - uv0.x, uv2.x - uv0.x};
  auto t   = vec2f{uv1.y - uv0.y, uv2.y - uv0.y};
  auto div = s.x * t.y - s.y * t.x;

  if (div != 0) {
    auto tu = vec3f{t.y * p.x - t.x * q.x, t.y * p.y - t.x * q.y,
                  t.y * p.z - t.x * q.z} /
              div;
    auto tv = vec3f{s.x * q.x - s.y * p.x, s.x * q.y - s.y * p.y,
                  s.x * q.z - s.y * p.z} /
              div;
    return {tu, tv};
  } else {
    return {{1, 0, 0}, {0, 1, 0}};
  }
}

// Quad tangent and bitangent from uv.
inline pair<vec3f, vec3f> quad_tangents_fromuv(const vec3f& p0, const vec3f& p1,
    const vec3f& p2, const vec3f& p3, const vec2f& uv0, const vec2f& uv1,
    const vec2f& uv2, const vec2f& uv3, const vec2f& current_uv) {
  if (current_uv.x + current_uv.y <= 1) {
    return triangle_tangents_fromuv(p0, p1, p3, uv0, uv1, uv3);
  } else {
    return triangle_tangents_fromuv(p2, p3, p1, uv2, uv3, uv1);
  }
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// RANDOM NUMBER GENERATION
// -----------------------------------------------------------------------------
namespace yocto {

// PCG random numbers from http://www.pcg-random.org/
struct rng_state {
  uint64_t state = 0x853c49e6748fea9bULL;
  uint64_t inc   = 0xda3e39cb94b95bdbULL;

  rng_state() = default;
  rng_state(uint64_t state, uint64_t inc);
};

// PCG random numbers from http://www.pcg-random.org/
inline rng_state::rng_state(uint64_t state, uint64_t inc)
    : state{state}, inc{inc} {}

// Next random number, used internally only.
inline uint32_t _advance_rng(rng_state& rng) {
  uint64_t oldstate = rng.state;
  rng.state         = oldstate * 6364136223846793005ULL + rng.inc;
  auto xorshifted   = (uint32_t)(((oldstate >> 18u) ^ oldstate) >> 27u);
  auto rot          = (uint32_t)(oldstate >> 59u);
  // return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
  return (xorshifted >> rot) | (xorshifted << ((~rot + 1u) & 31));
}

// Init a random number generator with a state state from the sequence seq.
inline rng_state make_rng(uint64_t seed, uint64_t seq) {
  auto rng  = rng_state();
  rng.state = 0U;
  rng.inc   = (seq << 1u) | 1u;
  _advance_rng(rng);
  rng.state += seed;
  _advance_rng(rng);
  return rng;
}

// Next random numbers: floats in [0,1), ints in [0,n).
inline int   rand1i(rng_state& rng, int n) { return _advance_rng(rng) % n; }
inline float rand1f(rng_state& rng) {
  union {
    uint32_t u;
    float    f;
  } x;
  x.u = (_advance_rng(rng) >> 9) | 0x3f800000u;
  return x.f - 1.0f;
  // alternate implementation
  // const static auto scale = (float)(1.0 / numeric_limits<uint32_t>::max());
  // return advance_rng(rng) * scale;
}
inline vec2f rand2f(rng_state& rng) {
  // force order of evaluation by using separate assignments.
  auto x = rand1f(rng);
  auto y = rand1f(rng);
  return {x, y};
}
inline vec3f rand3f(rng_state& rng) {
  // force order of evaluation by using separate assignments.
  auto x = rand1f(rng);
  auto y = rand1f(rng);
  auto z = rand1f(rng);
  return {x, y, z};
}

// Sample an hemispherical direction with uniform distribution.
inline vec3f sample_hemisphere(const vec2f& ruv) {
  auto z   = ruv.y;
  auto r   = sqrt(clamp(1 - z * z, 0.0f, 1.0f));
  auto phi = 2 * pif * ruv.x;
  return {r * cos(phi), r * sin(phi), z};
}
inline float sample_hemisphere_pdf(const vec3f& direction) {
  return (direction.z <= 0) ? 0 : 1 / (2 * pif);
}

// Sample an hemispherical direction with uniform distribution.
inline vec3f sample_hemisphere(const vec3f& normal, const vec2f& ruv) {
  auto z               = ruv.y;
  auto r               = sqrt(clamp(1 - z * z, 0.0f, 1.0f));
  auto phi             = 2 * pif * ruv.x;
  auto local_direction = vec3f{r * cos(phi), r * sin(phi), z};
  return transform_direction(frame_fromz({0, 0, 0}, normal), local_direction);
}
inline float sample_hemisphere_pdf(
    const vec3f& normal, const vec3f& direction) {
  return (dot(normal, direction) <= 0) ? 0 : 1 / (2 * pif);
}

// Sample a spherical direction with uniform distribution.
inline vec3f sample_sphere(const vec2f& ruv) {
  auto z   = 2 * ruv.y - 1;
  auto r   = sqrt(clamp(1 - z * z, 0.0f, 1.0f));
  auto phi = 2 * pif * ruv.x;
  return {r * cos(phi), r * sin(phi), z};
}
inline float sample_sphere_pdf(const vec3f& w) { return 1 / (4 * pif); }

// Sample an hemispherical direction with cosine distribution.
inline vec3f sample_hemisphere_cos(const vec2f& ruv) {
  auto z   = sqrt(ruv.y);
  auto r   = sqrt(1 - z * z);
  auto phi = 2 * pif * ruv.x;
  return {r * cos(phi), r * sin(phi), z};
}
inline float sample_hemisphere_cos_pdf(const vec3f& direction) {
  return (direction.z <= 0) ? 0 : direction.z / pif;
}

// Sample an hemispherical direction with cosine distribution.
inline vec3f sample_hemisphere_cos(const vec3f& normal, const vec2f& ruv) {
  auto z               = sqrt(ruv.y);
  auto r               = sqrt(1 - z * z);
  auto phi             = 2 * pif * ruv.x;
  auto local_direction = vec3f{r * cos(phi), r * sin(phi), z};
  return transform_direction(frame_fromz({0, 0, 0}, normal), local_direction);
}
inline float sample_hemisphere_cos_pdf(
    const vec3f& normal, const vec3f& direction) {
  auto cosw = dot(normal, direction);
  return (cosw <= 0) ? 0 : cosw / pif;
}

// Sample an hemispherical direction with cosine power distribution.
inline vec3f sample_hemisphere_cospower(float exponent, const vec2f& ruv) {
  auto z   = pow(ruv.y, 1 / (exponent + 1));
  auto r   = sqrt(1 - z * z);
  auto phi = 2 * pif * ruv.x;
  return {r * cos(phi), r * sin(phi), z};
}
inline float sample_hemisphere_cospower_pdf(
    float exponent, const vec3f& direction) {
  return (direction.z <= 0)
             ? 0
             : pow(direction.z, exponent) * (exponent + 1) / (2 * pif);
}

// Sample an hemispherical direction with cosine power distribution.
inline vec3f sample_hemisphere_cospower(
    float exponent, const vec3f& normal, const vec2f& ruv) {
  auto z               = pow(ruv.y, 1 / (exponent + 1));
  auto r               = sqrt(1 - z * z);
  auto phi             = 2 * pif * ruv.x;
  auto local_direction = vec3f{r * cos(phi), r * sin(phi), z};
  return transform_direction(frame_fromz({0, 0, 0}, normal), local_direction);
}
inline float sample_hemisphere_cospower_pdf(
    float exponent, const vec3f& normal, const vec3f& direction) {
  auto cosw = dot(normal, direction);
  return (cosw <= 0) ? 0 : pow(cosw, exponent) * (exponent + 1) / (2 * pif);
}

// Sample a point uniformly on a disk.
inline vec2f sample_disk(const vec2f& ruv) {
  auto r   = sqrt(ruv.y);
  auto phi = 2 * pif * ruv.x;
  return {cos(phi) * r, sin(phi) * r};
}
inline float sample_disk_pdf() { return 1 / pif; }

// Sample a point uniformly on a cylinder, without caps.
inline vec3f sample_cylinder(const vec2f& ruv) {
  auto phi = 2 * pif * ruv.x;
  return {sin(phi), cos(phi), ruv.y * 2 - 1};
}
inline float sample_cylinder_pdf(const vec3f& point) { return 1 / pif; }

// Sample a point uniformly on a triangle returning the baricentric coordinates.
inline vec2f sample_triangle(const vec2f& ruv) {
  return {1 - sqrt(ruv.x), ruv.y * sqrt(ruv.x)};
}

// Sample a point uniformly on a triangle.
inline vec3f sample_triangle(
    const vec3f& p0, const vec3f& p1, const vec3f& p2, const vec2f& ruv) {
  auto uv = sample_triangle(ruv);
  return p0 * (1 - uv.x - uv.y) + p1 * uv.x + p2 * uv.y;
}
// Pdf for uniform triangle sampling, i.e. triangle area.
inline float sample_triangle_pdf(
    const vec3f& p0, const vec3f& p1, const vec3f& p2) {
  return 2 / length(cross(p1 - p0, p2 - p0));
}

// Sample an index with uniform distribution.
inline int sample_uniform(int size, float r) {
  return clamp((int)(r * size), 0, size - 1);
}
inline float sample_uniform_pdf(int size) { return (float)1 / (float)size; }

// Sample an index with uniform distribution.
inline float sample_uniform(const span<float>& elements, float r) {
  if (elements.empty()) return {};
  auto size = (int)elements.size();
  return elements[clamp((int)(r * size), 0, size - 1)];
}
inline float sample_uniform_pdf(const span<float>& elements) {
  if (elements.empty()) return 0;
  return 1.0f / (int)elements.size();
}

// simplified version of possiblr implementation from cpprenference.com
template <class T>
static const T* _upper_bound(const T* first, const T* last, const T& value) {
  const T*  it;
  ptrdiff_t count, step;
  count = first - last;

  while (count > 0) {
    it   = first;
    step = count / 2;
    it += step;
    if (!(value < *it)) {
      first = ++it;
      count -= step + 1;
    } else
      count = step;
  }
  return first;
}

// Sample a discrete distribution represented by its cdf.
inline int sample_discrete(const span<float>& cdf, float r) {
  r = clamp(r * cdf.back(), (float)0, cdf.back() - (float)0.00001);
  auto idx =
      (int)(_upper_bound(cdf.data(), cdf.data() + cdf.size(), r) - cdf.data());
  return clamp(idx, 0, (int)cdf.size() - 1);
}
// Pdf for uniform discrete distribution sampling.
inline float sample_discrete_pdf(const span<float>& cdf, int idx) {
  if (idx == 0) return cdf.at(0);
  return cdf.at(idx) - cdf.at(idx - 1);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// SHADING FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Check if on the same side of the hemisphere
inline bool same_hemisphere(
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  return dot(normal, outgoing) * dot(normal, incoming) >= 0;
}

// Schlick approximation of the Fresnel term
inline vec3f fresnel_schlick(
    const vec3f& specular, const vec3f& normal, const vec3f& outgoing) {
  if (specular == vec3f{0, 0, 0}) return {0, 0, 0};
  auto cosine = dot(normal, outgoing);
  return specular +
         (1 - specular) * pow(clamp(1 - abs(cosine), 0.0f, 1.0f), 5.0f);
}

// Compute the fresnel term for dielectrics.
inline float fresnel_dielectric(
    float eta, const vec3f& normal, const vec3f& outgoing) {
  // Implementation from
  // https://seblagarde.wordpress.com/2013/04/29/memo-on-fresnel-equations/
  auto cosw = abs(dot(normal, outgoing));

  auto sin2 = 1 - cosw * cosw;
  auto eta2 = eta * eta;

  auto cos2t = 1 - sin2 / eta2;
  if (cos2t < 0) return 1;  // tir

  auto t0 = sqrt(cos2t);
  auto t1 = eta * t0;
  auto t2 = eta * cosw;

  auto rs = (cosw - t1) / (cosw + t1);
  auto rp = (t0 - t2) / (t0 + t2);

  return (rs * rs + rp * rp) / 2;
}

// Compute the fresnel term for metals.
inline vec3f fresnel_conductor(const vec3f& eta, const vec3f& etak,
    const vec3f& normal, const vec3f& outgoing) {
  // Implementation from
  // https://seblagarde.wordpress.com/2013/04/29/memo-on-fresnel-equations/
  auto cosw = dot(normal, outgoing);
  if (cosw <= 0) return {0, 0, 0};

  cosw       = clamp(cosw, (float)-1, (float)1);
  auto cos2  = cosw * cosw;
  auto sin2  = clamp(1 - cos2, (float)0, (float)1);
  auto eta2  = eta * eta;
  auto etak2 = etak * etak;

  auto t0       = eta2 - etak2 - sin2;
  auto a2plusb2 = sqrt(t0 * t0 + 4 * eta2 * etak2);
  auto t1       = a2plusb2 + cos2;
  auto a        = sqrt((a2plusb2 + t0) / 2);
  auto t2       = 2 * a * cosw;
  auto rs       = (t1 - t2) / (t1 + t2);

  auto t3 = cos2 * a2plusb2 + sin2 * sin2;
  auto t4 = t2 * sin2;
  auto rp = rs * (t3 - t4) / (t3 + t4);

  return (rp + rs) / 2;
}

// Convert eta to reflectivity
inline vec3f eta_to_reflectivity(const vec3f& eta) {
  return ((eta - 1) * (eta - 1)) / ((eta + 1) * (eta + 1));
}
// Convert reflectivity to  eta.
inline vec3f reflectivity_to_eta(const vec3f& reflectivity_) {
  auto reflectivity = clamp(reflectivity_, 0.0f, 0.99f);
  return (1 + sqrt(reflectivity)) / (1 - sqrt(reflectivity));
}
// Convert conductor eta to reflectivity
inline vec3f eta_to_reflectivity(const vec3f& eta, const vec3f& etak) {
  return ((eta - 1) * (eta - 1) + etak * etak) /
         ((eta + 1) * (eta + 1) + etak * etak);
}
// Convert eta to edge tint parametrization
inline pair<vec3f, vec3f> eta_to_edgetint(const vec3f& eta, const vec3f& etak) {
  auto reflectivity = eta_to_reflectivity(eta, etak);
  auto numer        = (1 + sqrt(reflectivity)) / (1 - sqrt(reflectivity)) - eta;
  auto denom        = (1 + sqrt(reflectivity)) / (1 - sqrt(reflectivity)) -
               (1 - reflectivity) / (1 + reflectivity);
  auto edgetint = numer / denom;
  return {reflectivity, edgetint};
}
// Convert reflectivity and edge tint to eta.
inline pair<vec3f, vec3f> edgetint_to_eta(
    const vec3f& reflectivity, const vec3f& edgetint) {
  auto r = clamp(reflectivity, 0.0f, 0.99f);
  auto g = edgetint;

  auto r_sqrt = sqrt(r);
  auto n_min  = (1 - r) / (1 + r);
  auto n_max  = (1 + r_sqrt) / (1 - r_sqrt);

  auto n  = lerp(n_max, n_min, g);
  auto k2 = ((n + 1) * (n + 1) * r - (n - 1) * (n - 1)) / (1 - r);
  k2      = max(k2, 0.0f);
  auto k  = sqrt(k2);
  return {n, k};
}

// Evaluate microfacet distribution
inline float microfacet_distribution(float roughness, const vec3f& normal,
    const vec3f& halfway, bool ggx = true) {
  // https://google.github.io/filament/Filament.html#materialsystem/specularbrdf
  // http://graphicrants.blogspot.com/2013/08/specular-brdf-reference.html
  auto cosine = dot(normal, halfway);
  if (cosine <= 0) return 0;
  auto roughness2 = roughness * roughness;
  auto cosine2    = cosine * cosine;
  if (ggx) {
    return roughness2 / (pif * (cosine2 * roughness2 + 1 - cosine2) *
                            (cosine2 * roughness2 + 1 - cosine2));
  } else {
    return exp((cosine2 - 1) / (roughness2 * cosine2)) /
           (pif * roughness2 * cosine2 * cosine2);
  }
}

// Evaluate the microfacet shadowing1
inline float microfacet_shadowing1(float roughness, const vec3f& normal,
    const vec3f& halfway, const vec3f& direction, bool ggx = true) {
  // https://google.github.io/filament/Filament.html#materialsystem/specularbrdf
  // http://graphicrants.blogspot.com/2013/08/specular-brdf-reference.html
  // https://github.com/KhronosGroup/glTF/tree/master/specification/2.0#appendix-b-brdf-implementation
  auto cosine  = dot(normal, direction);
  auto cosineh = dot(halfway, direction);
  if (cosine * cosineh <= 0) return 0;
  auto roughness2 = roughness * roughness;
  auto cosine2    = cosine * cosine;
  if (ggx) {
    return 2 * abs(cosine) /
           (abs(cosine) + sqrt(cosine2 - roughness2 * cosine2 + roughness2));
  } else {
    auto ci = abs(cosine) / (roughness * sqrt(1 - cosine2));
    return ci < 1.6f ? (3.535f * ci + 2.181f * ci * ci) /
                           (1.0f + 2.276f * ci + 2.577f * ci * ci)
                     : 1.0f;
  }
}

// Evaluate microfacet shadowing
inline float microfacet_shadowing(float roughness, const vec3f& normal,
    const vec3f& halfway, const vec3f& outgoing, const vec3f& incoming,
    bool ggx = true) {
  return microfacet_shadowing1(roughness, normal, halfway, outgoing, ggx) *
         microfacet_shadowing1(roughness, normal, halfway, incoming, ggx);
}

// Sample a microfacet distribution.
inline vec3f sample_microfacet(
    float roughness, const vec3f& normal, const vec2f& rn, bool ggx = true) {
  auto phi   = 2 * pif * rn.x;
  auto theta = 0.0f;
  if (ggx) {
    theta = atan(roughness * sqrt(rn.y / (1 - rn.y)));
  } else {
    auto roughness2 = roughness * roughness;
    theta           = atan(sqrt(-roughness2 * log(1 - rn.y)));
  }
  auto local_half_vector = vec3f{
      cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta)};
  return transform_direction(frame_fromz({0, 0, 0}, normal), local_half_vector);
}

// Pdf for microfacet distribution sampling.
inline float sample_microfacet_pdf(float roughness, const vec3f& normal,
    const vec3f& halfway, bool ggx = true) {
  auto cosine = dot(normal, halfway);
  if (cosine < 0) return 0;
  return microfacet_distribution(roughness, normal, halfway, ggx) * cosine;
}

// Pdf for microfacet distribution sampling with the distribution of visible
// normals.
inline float sample_microfacet_pdf(float roughness, const vec3f& normal,
    const vec3f& halfway, const vec3f& outgoing, bool ggx = true) {
  // http://jcgt.org/published/0007/04/01/
  if (dot(normal, halfway) < 0) return 0;
  if (dot(halfway, outgoing) < 0) return 0;
  return microfacet_distribution(roughness, normal, halfway, ggx) *
         microfacet_shadowing1(roughness, normal, halfway, outgoing, ggx) *
         max(0.0f, dot(halfway, outgoing)) / abs(dot(normal, outgoing));
}

// Microfacet energy compensation (E(cos(w)))
inline float microfacet_cosintegral(
    float roughness, const vec3f& normal, const vec3f& outgoing) {
  // https://blog.selfshadow.com/publications/s2017-shading-course/imageworks/s2017_pbs_imageworks_slides_v2.pdf
  const float S[5] = {-0.170718f, 4.07985f, -11.5295f, 18.4961f, -9.23618f};
  const float T[5] = {0.0632331f, 3.1434f, -7.47567f, 13.0482f, -7.0401f};
  auto        m    = abs(dot(normal, outgoing));
  auto        r    = roughness;
  auto        s = S[0] * sqrt(m) + S[1] * r + S[2] * r * r + S[3] * r * r * r +
           S[4] * r * r * r * r;
  auto t = T[0] * m + T[1] * r + T[2] * r * r + T[3] * r * r * r +
           T[4] * r * r * r * r;
  return 1 - pow(s, 6.0f) * pow(m, 3.0f / 4.0f) / (pow(t, 6.0f) + pow(m, 2.0f));
}
// Approximate microfacet compensation for metals with Schlick's Fresnel
inline vec3f microfacet_compensation(const vec3f& color, float roughness,
    const vec3f& normal, const vec3f& outgoing) {
  // https://blog.selfshadow.com/publications/turquin/ms_comp_final.pdf
  auto E = microfacet_cosintegral(sqrt(roughness), normal, outgoing);
  return 1 + color * (1 - E) / E;
}

// Evaluate a diffuse BRDF lobe.
inline vec3f eval_matte(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  return color / pif * abs(dot(normal, incoming));
}

// Sample a diffuse BRDF lobe.
inline vec3f sample_matte(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec2f& rn) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return sample_hemisphere_cos(up_normal, rn);
}

// Pdf for diffuse BRDF lobe sampling.
inline float sample_matte_pdf(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return sample_hemisphere_cos_pdf(up_normal, incoming);
}

// Evaluate a specular BRDF lobe.
inline vec3f eval_glossy(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto F1        = fresnel_dielectric(ior, up_normal, outgoing);
  auto halfway   = normalize(incoming + outgoing);
  auto F         = fresnel_dielectric(ior, halfway, incoming);
  auto D         = microfacet_distribution(roughness, up_normal, halfway);
  auto G         = microfacet_shadowing(
              roughness, up_normal, halfway, outgoing, incoming);
  return color * (1 - F1) / pif * abs(dot(up_normal, incoming)) +
         vec3f{1, 1, 1} * F * D * G /
             (4 * dot(up_normal, outgoing) * dot(up_normal, incoming)) *
             abs(dot(up_normal, incoming));
}

// Sample a specular BRDF lobe.
inline vec3f sample_glossy(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, float rnl, const vec2f& rn) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  if (rnl < fresnel_dielectric(ior, up_normal, outgoing)) {
    auto halfway  = sample_microfacet(roughness, up_normal, rn);
    auto incoming = reflect(outgoing, halfway);
    if (!same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
    return incoming;
  } else {
    return sample_hemisphere_cos(up_normal, rn);
  }
}

// Pdf for specular BRDF lobe sampling.
inline float sample_glossy_pdf(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = normalize(outgoing + incoming);
  auto F         = fresnel_dielectric(ior, up_normal, outgoing);
  return F * sample_microfacet_pdf(roughness, up_normal, halfway) /
             (4 * abs(dot(outgoing, halfway))) +
         (1 - F) * sample_hemisphere_cos_pdf(up_normal, incoming);
}

// Evaluate a metal BRDF lobe.
inline vec3f eval_reflective(const vec3f& color, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = normalize(incoming + outgoing);
  auto F         = fresnel_conductor(
              reflectivity_to_eta(color), {0, 0, 0}, halfway, incoming);
  auto D = microfacet_distribution(roughness, up_normal, halfway);
  auto G = microfacet_shadowing(
      roughness, up_normal, halfway, outgoing, incoming);
  return F * D * G / (4 * dot(up_normal, outgoing) * dot(up_normal, incoming)) *
         abs(dot(up_normal, incoming));
}

// Sample a metal BRDF lobe.
inline vec3f sample_reflective(const vec3f& color, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec2f& rn) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = sample_microfacet(roughness, up_normal, rn);
  auto incoming  = reflect(outgoing, halfway);
  if (!same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
  return incoming;
}

// Pdf for metal BRDF lobe sampling.
inline float sample_reflective_pdf(const vec3f& color, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = normalize(outgoing + incoming);
  return sample_microfacet_pdf(roughness, up_normal, halfway) /
         (4 * abs(dot(outgoing, halfway)));
}

// Evaluate a metal BRDF lobe.
inline vec3f eval_reflective(const vec3f& eta, const vec3f& etak,
    float roughness, const vec3f& normal, const vec3f& outgoing,
    const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = normalize(incoming + outgoing);
  auto F         = fresnel_conductor(eta, etak, halfway, incoming);
  auto D         = microfacet_distribution(roughness, up_normal, halfway);
  auto G         = microfacet_shadowing(
              roughness, up_normal, halfway, outgoing, incoming);
  return F * D * G / (4 * dot(up_normal, outgoing) * dot(up_normal, incoming)) *
         abs(dot(up_normal, incoming));
}

// Sample a metal BRDF lobe.
inline vec3f sample_reflective(const vec3f& eta, const vec3f& etak,
    float roughness, const vec3f& normal, const vec3f& outgoing,
    const vec2f& rn) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = sample_microfacet(roughness, up_normal, rn);
  return reflect(outgoing, halfway);
}

// Pdf for metal BRDF lobe sampling.
inline float sample_reflective_pdf(const vec3f& eta, const vec3f& etak,
    float roughness, const vec3f& normal, const vec3f& outgoing,
    const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = normalize(outgoing + incoming);
  return sample_microfacet_pdf(roughness, up_normal, halfway) /
         (4 * abs(dot(outgoing, halfway)));
}

// Evaluate a delta metal BRDF lobe.
inline vec3f eval_reflective(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return fresnel_conductor(
      reflectivity_to_eta(color), {0, 0, 0}, up_normal, outgoing);
}

// Sample a delta metal BRDF lobe.
inline vec3f sample_reflective(
    const vec3f& color, const vec3f& normal, const vec3f& outgoing) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return reflect(outgoing, up_normal);
}

// Pdf for delta metal BRDF lobe sampling.
inline float sample_reflective_pdf(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  return 1;
}

// Evaluate a delta metal BRDF lobe.
inline vec3f eval_reflective(const vec3f& eta, const vec3f& etak,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return fresnel_conductor(eta, etak, up_normal, outgoing);
}

// Sample a delta metal BRDF lobe.
inline vec3f sample_reflective(const vec3f& eta, const vec3f& etak,
    const vec3f& normal, const vec3f& outgoing) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return reflect(outgoing, up_normal);
}

// Pdf for delta metal BRDF lobe sampling.
inline float sample_reflective_pdf(const vec3f& eta, const vec3f& etak,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  return 1;
}

// Evaluate a specular BRDF lobe.
inline vec3f eval_gltfpbr(const vec3f& color, float ior, float roughness,
    float metallic, const vec3f& normal, const vec3f& outgoing,
    const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return {0, 0, 0};
  auto reflectivity = lerp(
      eta_to_reflectivity(vec3f{ior, ior, ior}), color, metallic);
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto F1        = fresnel_schlick(reflectivity, up_normal, outgoing);
  auto halfway   = normalize(incoming + outgoing);
  auto F         = fresnel_schlick(reflectivity, halfway, incoming);
  auto D         = microfacet_distribution(roughness, up_normal, halfway);
  auto G         = microfacet_shadowing(
              roughness, up_normal, halfway, outgoing, incoming);
  return color * (1 - metallic) * (1 - F1) / pif *
             abs(dot(up_normal, incoming)) +
         F * D * G / (4 * dot(up_normal, outgoing) * dot(up_normal, incoming)) *
             abs(dot(up_normal, incoming));
}

// Sample a specular BRDF lobe.
inline vec3f sample_gltfpbr(const vec3f& color, float ior, float roughness,
    float metallic, const vec3f& normal, const vec3f& outgoing, float rnl,
    const vec2f& rn) {
  auto up_normal    = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto reflectivity = lerp(
      eta_to_reflectivity(vec3f{ior, ior, ior}), color, metallic);
  if (rnl < mean(fresnel_schlick(reflectivity, up_normal, outgoing))) {
    auto halfway  = sample_microfacet(roughness, up_normal, rn);
    auto incoming = reflect(outgoing, halfway);
    if (!same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
    return incoming;
  } else {
    return sample_hemisphere_cos(up_normal, rn);
  }
}

// Pdf for specular BRDF lobe sampling.
inline float sample_gltfpbr_pdf(const vec3f& color, float ior, float roughness,
    float metallic, const vec3f& normal, const vec3f& outgoing,
    const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) <= 0) return 0;
  auto up_normal    = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway      = normalize(outgoing + incoming);
  auto reflectivity = lerp(
      eta_to_reflectivity(vec3f{ior, ior, ior}), color, metallic);
  auto F = mean(fresnel_schlick(reflectivity, up_normal, outgoing));
  return F * sample_microfacet_pdf(roughness, up_normal, halfway) /
             (4 * abs(dot(outgoing, halfway))) +
         (1 - F) * sample_hemisphere_cos_pdf(up_normal, incoming);
}

// Evaluate a transmission BRDF lobe.
inline vec3f eval_transparent(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    auto halfway = normalize(incoming + outgoing);
    auto F       = fresnel_dielectric(ior, halfway, outgoing);
    auto D       = microfacet_distribution(roughness, up_normal, halfway);
    auto G       = microfacet_shadowing(
              roughness, up_normal, halfway, outgoing, incoming);
    return vec3f{1, 1, 1} * F * D * G /
           (4 * dot(up_normal, outgoing) * dot(up_normal, incoming)) *
           abs(dot(up_normal, incoming));
  } else {
    auto reflected = reflect(-incoming, up_normal);
    auto halfway   = normalize(reflected + outgoing);
    auto F         = fresnel_dielectric(ior, halfway, outgoing);
    auto D         = microfacet_distribution(roughness, up_normal, halfway);
    auto G         = microfacet_shadowing(
                roughness, up_normal, halfway, outgoing, reflected);
    return color * (1 - F) * D * G /
           (4 * dot(up_normal, outgoing) * dot(up_normal, reflected)) *
           (abs(dot(up_normal, reflected)));
  }
}

// Sample a transmission BRDF lobe.
inline vec3f sample_transparent(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, float rnl, const vec2f& rn) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  auto halfway   = sample_microfacet(roughness, up_normal, rn);
  if (rnl < fresnel_dielectric(ior, halfway, outgoing)) {
    auto incoming = reflect(outgoing, halfway);
    if (!same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
    return incoming;
  } else {
    auto reflected = reflect(outgoing, halfway);
    auto incoming  = -reflect(reflected, up_normal);
    if (same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
    return incoming;
  }
}

// Pdf for transmission BRDF lobe sampling.
inline float sample_tranparent_pdf(const vec3f& color, float ior,
    float roughness, const vec3f& normal, const vec3f& outgoing,
    const vec3f& incoming) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    auto halfway = normalize(incoming + outgoing);
    return fresnel_dielectric(ior, halfway, outgoing) *
           sample_microfacet_pdf(roughness, up_normal, halfway) /
           (4 * abs(dot(outgoing, halfway)));
  } else {
    auto reflected = reflect(-incoming, up_normal);
    auto halfway   = normalize(reflected + outgoing);
    auto d         = (1 - fresnel_dielectric(ior, halfway, outgoing)) *
             sample_microfacet_pdf(roughness, up_normal, halfway);
    return d / (4 * abs(dot(outgoing, halfway)));
  }
}

// Evaluate a delta transmission BRDF lobe.
inline vec3f eval_transparent(const vec3f& color, float ior,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    return vec3f{1, 1, 1} * fresnel_dielectric(ior, up_normal, outgoing);
  } else {
    return color * (1 - fresnel_dielectric(ior, up_normal, outgoing));
  }
}

// Sample a delta transmission BRDF lobe.
inline vec3f sample_transparent(const vec3f& color, float ior,
    const vec3f& normal, const vec3f& outgoing, float rnl) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  if (rnl < fresnel_dielectric(ior, up_normal, outgoing)) {
    return reflect(outgoing, up_normal);
  } else {
    return -outgoing;
  }
}

// Pdf for delta transmission BRDF lobe sampling.
inline float sample_tranparent_pdf(const vec3f& color, float ior,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    return fresnel_dielectric(ior, up_normal, outgoing);
  } else {
    return 1 - fresnel_dielectric(ior, up_normal, outgoing);
  }
}

// Evaluate a refraction BRDF lobe.
inline vec3f eval_refractive(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  auto entering  = dot(normal, outgoing) >= 0;
  auto up_normal = entering ? normal : -normal;
  auto rel_ior   = entering ? ior : (1 / ior);
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    auto halfway = normalize(incoming + outgoing);
    auto F       = fresnel_dielectric(rel_ior, halfway, outgoing);
    auto D       = microfacet_distribution(roughness, up_normal, halfway);
    auto G       = microfacet_shadowing(
              roughness, up_normal, halfway, outgoing, incoming);
    return vec3f{1, 1, 1} * F * D * G /
           abs(4 * dot(normal, outgoing) * dot(normal, incoming)) *
           abs(dot(normal, incoming));
  } else {
    auto halfway = -normalize(rel_ior * incoming + outgoing) *
                   (entering ? 1.0f : -1.0f);
    auto F = fresnel_dielectric(rel_ior, halfway, outgoing);
    auto D = microfacet_distribution(roughness, up_normal, halfway);
    auto G = microfacet_shadowing(
        roughness, up_normal, halfway, outgoing, incoming);
    // [Walter 2007] equation 21
    return vec3f{1, 1, 1} *
           abs((dot(outgoing, halfway) * dot(incoming, halfway)) /
               (dot(outgoing, normal) * dot(incoming, normal))) *
           (1 - F) * D * G /
           pow(rel_ior * dot(halfway, incoming) + dot(halfway, outgoing),
               2.0f) *
           abs(dot(normal, incoming));
  }
}

// Sample a refraction BRDF lobe.
inline vec3f sample_refractive(const vec3f& color, float ior, float roughness,
    const vec3f& normal, const vec3f& outgoing, float rnl, const vec2f& rn) {
  auto entering  = dot(normal, outgoing) >= 0;
  auto up_normal = entering ? normal : -normal;
  auto halfway   = sample_microfacet(roughness, up_normal, rn);
  // auto halfway = sample_microfacet(roughness, up_normal, outgoing, rn);
  if (rnl < fresnel_dielectric(entering ? ior : (1 / ior), halfway, outgoing)) {
    auto incoming = reflect(outgoing, halfway);
    if (!same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
    return incoming;
  } else {
    auto incoming = refract(outgoing, halfway, entering ? (1 / ior) : ior);
    if (same_hemisphere(up_normal, outgoing, incoming)) return {0, 0, 0};
    return incoming;
  }
}

// Pdf for refraction BRDF lobe sampling.
inline float sample_refractive_pdf(const vec3f& color, float ior,
    float roughness, const vec3f& normal, const vec3f& outgoing,
    const vec3f& incoming) {
  auto entering  = dot(normal, outgoing) >= 0;
  auto up_normal = entering ? normal : -normal;
  auto rel_ior   = entering ? ior : (1 / ior);
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    auto halfway = normalize(incoming + outgoing);
    return fresnel_dielectric(rel_ior, halfway, outgoing) *
           sample_microfacet_pdf(roughness, up_normal, halfway) /
           //  sample_microfacet_pdf(roughness, up_normal, halfway, outgoing) /
           (4 * abs(dot(outgoing, halfway)));
  } else {
    auto halfway = -normalize(rel_ior * incoming + outgoing) *
                   (entering ? 1.0f : -1.0f);
    // [Walter 2007] equation 17
    return (1 - fresnel_dielectric(rel_ior, halfway, outgoing)) *
           sample_microfacet_pdf(roughness, up_normal, halfway) *
           //  sample_microfacet_pdf(roughness, up_normal, halfway, outgoing) /
           abs(dot(halfway, incoming)) /  // here we use incoming as from pbrt
           pow(rel_ior * dot(halfway, incoming) + dot(halfway, outgoing), 2.0f);
  }
}

// Evaluate a delta refraction BRDF lobe.
inline vec3f eval_refractive(const vec3f& color, float ior, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (abs(ior - 1) < 1e-3)
    return dot(normal, incoming) * dot(normal, outgoing) <= 0 ? vec3f{1, 1, 1}
                                                              : vec3f{0, 0, 0};
  auto entering  = dot(normal, outgoing) >= 0;
  auto up_normal = entering ? normal : -normal;
  auto rel_ior   = entering ? ior : (1 / ior);
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    return vec3f{1, 1, 1} * fresnel_dielectric(rel_ior, up_normal, outgoing);
  } else {
    return vec3f{1, 1, 1} * (1 / (rel_ior * rel_ior)) *
           (1 - fresnel_dielectric(rel_ior, up_normal, outgoing));
  }
}

// Sample a delta refraction BRDF lobe.
inline vec3f sample_refractive(const vec3f& color, float ior,
    const vec3f& normal, const vec3f& outgoing, float rnl) {
  if (abs(ior - 1) < 1e-3) return -outgoing;
  auto entering  = dot(normal, outgoing) >= 0;
  auto up_normal = entering ? normal : -normal;
  auto rel_ior   = entering ? ior : (1 / ior);
  if (rnl < fresnel_dielectric(rel_ior, up_normal, outgoing)) {
    return reflect(outgoing, up_normal);
  } else {
    return refract(outgoing, up_normal, 1 / rel_ior);
  }
}

// Pdf for delta refraction BRDF lobe sampling.
inline float sample_refractive_pdf(const vec3f& color, float ior,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (abs(ior - 1) < 1e-3)
    return dot(normal, incoming) * dot(normal, outgoing) < 0 ? 1.0f : 0.0f;
  auto entering  = dot(normal, outgoing) >= 0;
  auto up_normal = entering ? normal : -normal;
  auto rel_ior   = entering ? ior : (1 / ior);
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    return fresnel_dielectric(rel_ior, up_normal, outgoing);
  } else {
    return (1 - fresnel_dielectric(rel_ior, up_normal, outgoing));
  }
}

// Evaluate a translucent BRDF lobe.
inline vec3f eval_translucent(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) return {0, 0, 0};
  return color / pif * abs(dot(normal, incoming));
}

// Sample a translucency BRDF lobe.
inline vec3f sample_translucent(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec2f& rn) {
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return sample_hemisphere_cos(-up_normal, rn);
}

// Pdf for translucency BRDF lobe sampling.
inline float sample_translucent_pdf(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) return 0;
  auto up_normal = dot(normal, outgoing) <= 0 ? -normal : normal;
  return sample_hemisphere_cos_pdf(-up_normal, incoming);
}

// Evaluate a passthrough BRDF lobe.
inline vec3f eval_passthrough(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    return vec3f{0, 0, 0};
  } else {
    return vec3f{1, 1, 1};
  }
}

// Sample a passthrough BRDF lobe.
inline vec3f sample_passthrough(
    const vec3f& color, const vec3f& normal, const vec3f& outgoing) {
  return -outgoing;
}

// Pdf for passthrough BRDF lobe sampling.
inline float sample_passthrough_pdf(const vec3f& color, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (dot(normal, incoming) * dot(normal, outgoing) >= 0) {
    return 0;
  } else {
    return 1;
  }
}

// Convert mean-free-path to transmission
inline vec3f mfp_to_transmission(const vec3f& mfp, float depth) {
  return exp(-depth / mfp);
}

// Evaluate transmittance
inline vec3f eval_transmittance(const vec3f& density, float distance) {
  return exp(-density * distance);
}

// Sample a distance proportionally to transmittance
inline float sample_transmittance(
    const vec3f& density, float max_distance, float rl, float rd) {
  auto channel  = clamp((int)(rl * 3), 0, 2);
  auto distance = (density[channel] == 0) ? flt_max
                                          : -log(1 - rd) / density[channel];
  return min(distance, max_distance);
}

// Pdf for distance sampling
inline float sample_transmittance_pdf(
    const vec3f& density, float distance, float max_distance) {
  if (distance < max_distance) {
    return sum(density * exp(-density * distance)) / 3;
  } else {
    return sum(exp(-density * max_distance)) / 3;
  }
}

// Evaluate phase function
inline float eval_phasefunction(
    float anisotropy, const vec3f& outgoing, const vec3f& incoming) {
  auto cosine = -dot(outgoing, incoming);
  auto denom  = 1 + anisotropy * anisotropy - 2 * anisotropy * cosine;
  return (1 - anisotropy * anisotropy) / (4 * pif * denom * sqrt(denom));
}

// Sample phase function
inline vec3f sample_phasefunction(
    float anisotropy, const vec3f& outgoing, const vec2f& rn) {
  auto cos_theta = 0.0f;
  if (abs(anisotropy) < 1e-3f) {
    cos_theta = 1 - 2 * rn.y;
  } else {
    auto square = (1 - anisotropy * anisotropy) /
                  (1 + anisotropy - 2 * anisotropy * rn.y);
    cos_theta = (1 + anisotropy * anisotropy - square * square) /
                (2 * anisotropy);
  }

  auto sin_theta      = sqrt(max(0.0f, 1 - cos_theta * cos_theta));
  auto phi            = 2 * pif * rn.x;
  auto local_incoming = vec3f{
      sin_theta * cos(phi), sin_theta * sin(phi), cos_theta};
  return transform_direction(frame_fromz({0, 0, 0}, -outgoing), local_incoming);
}

// Pdf for phase function sampling
inline float sample_phasefunction_pdf(
    float anisotropy, const vec3f& outgoing, const vec3f& incoming) {
  return eval_phasefunction(anisotropy, outgoing, incoming);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// CUDA HELPERS
// -----------------------------------------------------------------------------
namespace yocto {

template <typename T>
struct cubuffer {
  inline bool     empty() const { return _size == 0; }
  inline size_t   size() const { return _size; }
  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline operator span<T>() const { return {_data, _size}; }

  T*     _data = nullptr;
  size_t _size = 0;
};

template <typename T, size_t Size = 16>
struct svector {
  inline bool     empty() const { return _size == 0; }
  inline size_t   size() const { return _size; }
  inline T&       operator[](int idx) { return _data[idx]; }
  inline const T& operator[](int idx) const { return _data[idx]; }

  inline T*       begin() { return _data; }
  inline T*       end() { return _data + _size; }
  inline const T* begin() const { return _data; }
  inline const T* end() const { return _data + _size; }

  inline T&       front() { return *_data; }
  inline T&       back() { return *(_data + _size - 1); }
  inline const T& front() const { return *_data; }
  inline const T& back() const { return *(_data + _size - 1); }

  inline void push_back(const T& value) { _data[_size++] = value; }
  inline void pop_back() { _size--; }

  T      _data[Size] = {};
  size_t _size       = 0;
};

inline void* unpackPointer(uint32_t i0, uint32_t i1) {
  const uint64_t uptr = static_cast<uint64_t>(i0) << 32 | i1;
  void*          ptr  = reinterpret_cast<void*>(uptr);
  return ptr;
}

inline void packPointer(void* ptr, uint32_t& i0, uint32_t& i1) {
  const uint64_t uptr = reinterpret_cast<uint64_t>(ptr);
  i0                  = uptr >> 32;
  i1                  = uptr & 0x00000000ffffffff;
}

template <typename T>
inline T* getPRD() {
  const uint32_t u0 = optixGetPayload_0();
  const uint32_t u1 = optixGetPayload_1();
  return reinterpret_cast<T*>(unpackPointer(u0, u1));
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// CUTRACE TYPES
// -----------------------------------------------------------------------------
namespace yocto {

constexpr int invalidid = -1;

struct cutrace_state {
  int                 width   = 0;
  int                 height  = 0;
  int                 samples = 0;
  cubuffer<vec4f>     image   = {};
  cubuffer<vec3f>     albedo  = {};
  cubuffer<vec3f>     normal  = {};
  cubuffer<int>       hits    = {};
  cubuffer<rng_state> rngs    = {};
  cubuffer<vec4f>     display = {};
};

struct cutrace_camera {
  frame3f frame        = {};
  float   lens         = {};
  float   film         = {};
  float   aspect       = {};
  float   focus        = {};
  float   aperture     = {};
  bool    orthographic = {};
};

struct cutrace_texture {
  hipArray_t         array   = nullptr;
  hipTextureObject_t texture = 0;
  int                 width   = 0;
  int                 height  = 0;
  bool                linear  = false;
};

enum struct material_type {
  // clang-format off
  matte, glossy, reflective, transparent, refractive, subsurface, volumetric, 
  gltfpbr
  // clang-format on
};

struct cutrace_material {
  material_type type         = material_type::matte;
  vec3f         emission     = {0, 0, 0};
  vec3f         color        = {0, 0, 0};
  float         roughness    = 0;
  float         metallic     = 0;
  float         ior          = 1.5f;
  vec3f         scattering   = {0, 0, 0};
  float         scanisotropy = 0;
  float         trdepth      = 0.01f;
  float         opacity      = 1;

  int emission_tex   = invalidid;
  int color_tex      = invalidid;
  int roughness_tex  = invalidid;
  int scattering_tex = invalidid;
  int normal_tex     = invalidid;
};

struct cutrace_instance {
  frame3f frame    = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {0, 0, 0}};
  int     shape    = -1;
  int     material = -1;
};

struct cutrace_shape {
  cubuffer<vec3f> positions = {};
  cubuffer<vec3f> normals   = {};
  cubuffer<vec2f> texcoords = {};
  cubuffer<vec4f> colors    = {};
  cubuffer<vec3i> triangles = {};
};

struct cutrace_environment {
  frame3f frame        = {{1, 0, 0}, {0, 1, 0}, {0, 0, 1}, {0, 0, 0}};
  vec3f   emission     = {0, 0, 0};
  int     emission_tex = invalidid;
};

struct cutrace_scene {
  cubuffer<cutrace_camera>      cameras      = {};
  cubuffer<cutrace_texture>     textures     = {};
  cubuffer<cutrace_material>    materials    = {};
  cubuffer<cutrace_shape>       shapes       = {};
  cubuffer<cutrace_instance>    instances    = {};
  cubuffer<cutrace_environment> environments = {};
};

// Type of tracing algorithm
enum struct cutrace_sampler_type {
  path,        // path tracing
  pathdirect,  // path tracing with direct
  pathmis,     // path tracing with mis
  naive,       // naive path tracing
  eyelight,    // eyelight rendering
  eyelightao,  // eyelight with ambient occlusion
  furnace,     // furnace test
  falsecolor,  // false color rendering
};
// Type of false color visualization
enum struct cutrace_falsecolor_type {
  // clang-format off
  position, normal, frontfacing, gnormal, gfrontfacing, texcoord, mtype, color,
  emission, roughness, opacity, metallic, delta, instance, shape, material, 
  element, highlight
  // clang-format on
};

// Default trace seed
constexpr auto cutrace_default_seed = 961748941ull;

// params
struct cutrace_params {
  int                     camera         = 0;
  int                     resolution     = 1280;
  cutrace_sampler_type    sampler        = cutrace_sampler_type::path;
  cutrace_falsecolor_type falsecolor     = cutrace_falsecolor_type::color;
  int                     samples        = 512;
  int                     bounces        = 8;
  float                   clamp          = 10;
  bool                    nocaustics     = false;
  bool                    envhidden      = false;
  bool                    tentfilter     = false;
  uint64_t                seed           = cutrace_default_seed;
  bool                    embreebvh      = false;
  bool                    highqualitybvh = false;
  bool                    noparallel     = false;
  int                     pratio         = 8;
  float                   exposure       = 0;
  bool                    filmic         = false;
  bool                    denoise        = false;
  int                     batch          = 1;
};

using cutrace_bvh = OptixTraversableHandle;

// light
struct cutrace_light {
  int             instance     = invalidid;
  int             environment  = invalidid;
  cubuffer<float> elements_cdf = {};
};

// lights
struct cutrace_lights {
  cubuffer<cutrace_light> lights = {};
};

struct cutrace_globals {
  cutrace_state          state  = {};
  cutrace_scene          scene  = {};
  OptixTraversableHandle bvh    = 0;
  cutrace_lights         lights = {};
  cutrace_params         params = {};
};

// global data
optix_constant cutrace_globals globals;

// compatibility aliases
using trace_bvh                   = cutrace_bvh;
using trace_lights                = cutrace_lights;
using trace_params                = cutrace_params;
using trace_falsecolor_type       = cutrace_falsecolor_type;
using trace_sampler_type          = cutrace_sampler_type;
constexpr auto trace_default_seed = cutrace_default_seed;

}  // namespace yocto

// -----------------------------------------------------------------------------
// SCENE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// compatibility aliases
using scene_data       = cutrace_scene;
using camera_data      = cutrace_camera;
using material_data    = cutrace_material;
using texture_data     = cutrace_texture;
using instance_data    = cutrace_instance;
using shape_data       = cutrace_shape;
using environment_data = cutrace_environment;

// constant values
constexpr auto min_roughness = 0.03f * 0.03f;

// Evaluates an image at a point `uv`.
static vec4f eval_texture(const texture_data& texture, const vec2f& texcoord,
    bool as_linear = false, bool no_interpolation = false,
    bool clamp_to_edge = false) {
  auto fromTexture = tex2D<float4>(texture.texture, texcoord.x, texcoord.y);
  auto color       = vec4f{
      fromTexture.x, fromTexture.y, fromTexture.z, fromTexture.w};
  if (as_linear && !texture.linear) {
    return srgb_to_rgb(color);
  } else {
    return color;
  }
}

// Helpers
static vec4f eval_texture(const scene_data& scene, int texture, const vec2f& uv,
    bool ldr_as_linear = false, bool no_interpolation = false,
    bool clamp_to_edge = false) {
  if (texture == invalidid) return {1, 1, 1, 1};
  return eval_texture(
      scene.textures[texture], uv, ldr_as_linear, no_interpolation);
}

// Material parameters evaluated at a point on the surface
struct material_point {
  material_type type         = material_type::gltfpbr;
  vec3f         emission     = {0, 0, 0};
  vec3f         color        = {0, 0, 0};
  float         opacity      = 1;
  float         roughness    = 0;
  float         metallic     = 0;
  float         ior          = 1;
  vec3f         density      = {0, 0, 0};
  vec3f         scattering   = {0, 0, 0};
  float         scanisotropy = 0;
  float         trdepth      = 0.01f;
};

// Evaluate material
static material_point eval_material(const scene_data& scene,
    const material_data& material, const vec2f& texcoord,
    const vec4f& color_shp) {
  // evaluate textures
  auto emission_tex = eval_texture(
      scene, material.emission_tex, texcoord, true);
  auto color_tex     = eval_texture(scene, material.color_tex, texcoord, true);
  auto roughness_tex = eval_texture(
      scene, material.roughness_tex, texcoord, false);
  auto scattering_tex = eval_texture(
      scene, material.scattering_tex, texcoord, true);

  // material point
  auto point         = material_point{};
  point.type         = material.type;
  point.emission     = material.emission * xyz(emission_tex);
  point.color        = material.color * xyz(color_tex) * xyz(color_shp);
  point.opacity      = material.opacity * color_tex.w * color_shp.w;
  point.metallic     = material.metallic * roughness_tex.z;
  point.roughness    = material.roughness * roughness_tex.y;
  point.roughness    = point.roughness * point.roughness;
  point.ior          = material.ior;
  point.scattering   = material.scattering * xyz(scattering_tex);
  point.scanisotropy = material.scanisotropy;
  point.trdepth      = material.trdepth;

  // volume density
  if (material.type == material_type::refractive ||
      material.type == material_type::volumetric ||
      material.type == material_type::subsurface) {
    point.density = -log(clamp(point.color, 0.0001f, 1.0f)) / point.trdepth;
  } else {
    point.density = {0, 0, 0};
  }

  // fix roughness
  if (point.type == material_type::matte ||
      point.type == material_type::gltfpbr ||
      point.type == material_type::glossy) {
    point.roughness = clamp(point.roughness, min_roughness, 1.0f);
  }

  return point;
}

// Eval position
static vec3f eval_position(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_point(
        instance.frame, interpolate_triangle(shape.positions[t.x],
                            shape.positions[t.y], shape.positions[t.z], uv));
  } else {
    return {0, 0, 0};
  }
}

// Shape element normal.
static vec3f eval_element_normal(
    const scene_data& scene, const instance_data& instance, int element) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_normal(
        instance.frame, triangle_normal(shape.positions[t.x],
                            shape.positions[t.y], shape.positions[t.z]));
  } else {
    return {0, 0, 0};
  }
}

// Eval normal
static vec3f eval_normal(const scene_data& scene, const instance_data& instance,
    int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.normals.empty())
    return eval_element_normal(scene, instance, element);
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return transform_normal(
        instance.frame, normalize(interpolate_triangle(shape.normals[t.x],
                            shape.normals[t.y], shape.normals[t.z], uv)));
  } else {
    return {0, 0, 0};
  }
}

// Eval texcoord
static vec2f eval_texcoord(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.texcoords.empty()) return uv;
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return interpolate_triangle(
        shape.texcoords[t.x], shape.texcoords[t.y], shape.texcoords[t.z], uv);
  } else {
    return {0, 0};
  }
}

// Shape element normal.
static pair<vec3f, vec3f> eval_element_tangents(
    const scene_data& scene, const instance_data& instance, int element) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty() && !shape.texcoords.empty()) {
    auto t   = shape.triangles[element];
    auto tuv = triangle_tangents_fromuv(shape.positions[t.x],
        shape.positions[t.y], shape.positions[t.z], shape.texcoords[t.x],
        shape.texcoords[t.y], shape.texcoords[t.z]);
    return {transform_direction(instance.frame, tuv.first),
        transform_direction(instance.frame, tuv.second)};
  } else {
    return {};
  }
}

static vec3f eval_normalmap(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& shape    = scene.shapes[instance.shape];
  auto& material = scene.materials[instance.material];
  // apply normal mapping
  auto normal   = eval_normal(scene, instance, element, uv);
  auto texcoord = eval_texcoord(scene, instance, element, uv);
  if (material.normal_tex != invalidid && (!shape.triangles.empty())) {
    auto& normal_tex = scene.textures[material.normal_tex];
    auto  normalmap  = -1 + 2 * xyz(eval_texture(normal_tex, texcoord, false));
    auto  tuv        = eval_element_tangents(scene, instance, element);
    auto  frame      = frame3f{tuv.first, tuv.second, normal, {0, 0, 0}};
    frame.x          = orthonormalize(frame.x, frame.z);
    frame.y          = normalize(cross(frame.z, frame.x));
    auto flip_v      = dot(frame.y, tuv.second) < 0;
    normalmap.y *= flip_v ? 1 : -1;  // flip vertical axis
    normal = transform_normal(frame, normalmap);
  }
  return normal;
}

// Eval shading position
static vec3f eval_shading_position(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv,
    const vec3f& outgoing) {
  auto& shape = scene.shapes[instance.shape];
  if (!shape.triangles.empty()) {
    return eval_position(scene, instance, element, uv);
  } else {
    return {0, 0, 0};
  }
}

// Eval shading normal
static vec3f eval_shading_normal(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv,
    const vec3f& outgoing) {
  auto& shape    = scene.shapes[instance.shape];
  auto& material = scene.materials[instance.material];
  if (!shape.triangles.empty()) {
    auto normal = eval_normal(scene, instance, element, uv);
    if (material.normal_tex != invalidid) {
      normal = eval_normalmap(scene, instance, element, uv);
    }
    if (material.type == material_type::refractive) return normal;
    return dot(normal, outgoing) >= 0 ? normal : -normal;
  } else {
    return {0, 0, 0};
  }
}

// Eval color
static vec4f eval_color(const scene_data& scene, const instance_data& instance,
    int element, const vec2f& uv) {
  auto& shape = scene.shapes[instance.shape];
  if (shape.colors.empty()) return {1, 1, 1, 1};
  if (!shape.triangles.empty()) {
    auto t = shape.triangles[element];
    return interpolate_triangle(
        shape.colors[t.x], shape.colors[t.y], shape.colors[t.z], uv);
  } else {
    return {0, 0, 0, 0};
  }
}

// Evaluate material
static material_point eval_material(const scene_data& scene,
    const instance_data& instance, int element, const vec2f& uv) {
  auto& material = scene.materials[instance.material];
  auto  texcoord = eval_texcoord(scene, instance, element, uv);

  // evaluate textures
  auto emission_tex = eval_texture(
      scene, material.emission_tex, texcoord, true);
  auto color_shp     = eval_color(scene, instance, element, uv);
  auto color_tex     = eval_texture(scene, material.color_tex, texcoord, true);
  auto roughness_tex = eval_texture(
      scene, material.roughness_tex, texcoord, false);
  auto scattering_tex = eval_texture(
      scene, material.scattering_tex, texcoord, true);

  // material point
  auto point         = material_point{};
  point.type         = material.type;
  point.emission     = material.emission * xyz(emission_tex);
  point.color        = material.color * xyz(color_tex) * xyz(color_shp);
  point.opacity      = material.opacity * color_tex.w * color_shp.w;
  point.metallic     = material.metallic * roughness_tex.z;
  point.roughness    = material.roughness * roughness_tex.y;
  point.roughness    = point.roughness * point.roughness;
  point.ior          = material.ior;
  point.scattering   = material.scattering * xyz(scattering_tex);
  point.scanisotropy = material.scanisotropy;
  point.trdepth      = material.trdepth;

  // volume density
  if (material.type == material_type::refractive ||
      material.type == material_type::volumetric ||
      material.type == material_type::subsurface) {
    point.density = -log(clamp(point.color, 0.0001f, 1.0f)) / point.trdepth;
  } else {
    point.density = {0, 0, 0};
  }

  // fix roughness
  if (point.type == material_type::matte ||
      point.type == material_type::gltfpbr ||
      point.type == material_type::glossy) {
    point.roughness = clamp(point.roughness, min_roughness, 1.0f);
  } else if (material.type == material_type::volumetric) {
    point.roughness = 0;
  } else {
    if (point.roughness < min_roughness) point.roughness = 0;
  }

  return point;
}

// check if a material is a delta or volumetric
static bool is_delta(const material_data& material) {
  return (material.type == material_type::reflective &&
             material.roughness == 0) ||
         (material.type == material_type::refractive &&
             material.roughness == 0) ||
         (material.type == material_type::transparent &&
             material.roughness == 0) ||
         (material.type == material_type::volumetric);
}
static bool is_volumetric(const material_data& material) {
  return material.type == material_type::refractive ||
         material.type == material_type::volumetric ||
         material.type == material_type::subsurface;
}

// check if an instance is volumetric
static bool is_volumetric(
    const scene_data& scene, const instance_data& instance) {
  return is_volumetric(scene.materials[instance.material]);
}

// check if a brdf is a delta
static bool is_delta(const material_point& material) {
  return (material.type == material_type::reflective &&
             material.roughness == 0) ||
         (material.type == material_type::refractive &&
             material.roughness == 0) ||
         (material.type == material_type::transparent &&
             material.roughness == 0) ||
         (material.type == material_type::volumetric);
}
static bool has_volume(const material_point& material) {
  return material.type == material_type::refractive ||
         material.type == material_type::volumetric ||
         material.type == material_type::subsurface;
}

static ray3f eval_camera(
    const cutrace_camera& camera, const vec2f& image_uv, const vec2f& lens_uv) {
  auto film = camera.aspect >= 1
                  ? vec2f{camera.film, camera.film / camera.aspect}
                  : vec2f{camera.film * camera.aspect, camera.film};
  auto q    = vec3f{
      film.x * (0.5f - image_uv.x), film.y * (image_uv.y - 0.5f), camera.lens};
  // ray direction through the lens center
  auto dc = -normalize(q);
  // point on the lens
  auto e = vec3f{
      lens_uv.x * camera.aperture / 2, lens_uv.y * camera.aperture / 2, 0};
  // point on the focus plane
  auto p = dc * camera.focus / abs(dc.z);
  // correct ray direction to account for camera focusing
  auto d = normalize(p - e);
  // done
  return ray3f{
      transform_point(camera.frame, e), transform_direction(camera.frame, d)};
}

// Evaluate environment color.
static vec3f eval_environment(const scene_data& scene,
    const environment_data& environment, const vec3f& direction) {
  auto wl       = transform_direction_inverse(environment.frame, direction);
  auto texcoord = vec2f{
      atan2(wl.z, wl.x) / (2 * pif), acos(clamp(wl.y, -1.0f, 1.0f)) / pif};
  if (texcoord.x < 0) texcoord.x += 1;
  return environment.emission *
         xyz(eval_texture(scene, environment.emission_tex, texcoord));
}

// Evaluate all environment color.
static vec3f eval_environment(const scene_data& scene, const vec3f& direction) {
  auto emission = vec3f{0, 0, 0};
  for (auto& environment : scene.environments) {
    emission += eval_environment(scene, environment, direction);
  }
  return emission;
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// RAY-SCENE INTERSECTION
// -----------------------------------------------------------------------------
namespace yocto {

// intersection result
struct scene_intersection {
  int   instance = -1;
  int   element  = -1;
  vec2f uv       = {0, 0};
  float distance = 0;
  bool  hit      = false;
  float _pad     = 0;
};

// closest hit
optix_shader void __closesthit__intersect_scene() {
  auto& intersection    = *getPRD<scene_intersection>();
  intersection.instance = optixGetInstanceIndex();
  intersection.element  = optixGetPrimitiveIndex();
  intersection.uv       = {
      optixGetTriangleBarycentrics().x, optixGetTriangleBarycentrics().y};
  intersection.distance = optixGetRayTmax();
  intersection.hit      = true;
}

// anyhit shader
optix_shader void __anyhit__intersect_scene() {}

// miss shader
optix_shader void __miss__intersect_scene() {
  auto& intersection    = *getPRD<scene_intersection>();
  intersection.instance = 0;
  intersection.element  = 0;
  intersection.uv       = {0, 0};
  intersection.distance = optixGetRayTmax();
  intersection.hit      = false;
}

// scene intersection via shaders
static scene_intersection intersect_scene(
    const trace_bvh& bvh, const cutrace_scene& scene, const ray3f& ray) {
  auto     intersection = scene_intersection{};
  uint32_t u0, u1;
  packPointer(&intersection, u0, u1);
  optixTrace(bvh, {ray.o.x, ray.o.y, ray.o.z}, {ray.d.x, ray.d.y, ray.d.z},
      ray.tmin, ray.tmax, 0.0f, OptixVisibilityMask(255),
      OPTIX_RAY_FLAG_DISABLE_ANYHIT, 0, 0, 0, u0, u1);
  return intersection;
}

// Primitive intersection
struct prim_intersection {
  vec2f uv       = {0, 0};
  float distance = flt_max;
  bool  hit      = false;
};

// Intersect a ray with a triangle
inline prim_intersection intersect_triangle(
    const ray3f& ray, const vec3f& p0, const vec3f& p1, const vec3f& p2) {
  // compute triangle edges
  auto edge1 = p1 - p0;
  auto edge2 = p2 - p0;

  // compute determinant to solve a linear system
  auto pvec = cross(ray.d, edge2);
  auto det  = dot(edge1, pvec);

  // check determinant and exit if triangle and ray are parallel
  // (could use EPSILONS if desired)
  if (det == 0) return {};
  auto inv_det = 1.0f / det;

  // compute and check first bricentric coordinated
  auto tvec = ray.o - p0;
  auto u    = dot(tvec, pvec) * inv_det;
  if (u < 0 || u > 1) return {};

  // compute and check second bricentric coordinated
  auto qvec = cross(tvec, edge1);
  auto v    = dot(ray.d, qvec) * inv_det;
  if (v < 0 || u + v > 1) return {};

  // compute and check ray parameter
  auto t = dot(edge2, qvec) * inv_det;
  if (t < ray.tmin || t > ray.tmax) return {};

  // intersection occurred: set params and exit
  return {{u, v}, t, true};
}

// instance intersection, for now manual
static scene_intersection intersect_instance(const trace_bvh& bvh,
    const cutrace_scene& scene, int instance_id, const ray3f& ray) {
  auto& instance     = scene.instances[instance_id];
  auto& shape        = scene.shapes[instance.shape];
  auto  intersection = scene_intersection{};
  auto  tray         = ray3f{transform_point_inverse(instance.frame, ray.o),
      transform_vector_inverse(instance.frame, ray.d)};
  for (auto element = 0; element < shape.triangles.size(); element++) {
    auto& triangle = shape.triangles[element];
    auto  isec     = intersect_triangle(tray, shape.positions[triangle.x],
             shape.positions[triangle.y], shape.positions[triangle.z]);
    if (!isec.hit) continue;
    intersection.hit      = true;
    intersection.instance = instance_id;
    intersection.element  = element;
    intersection.uv       = isec.uv;
    intersection.distance = isec.distance;
    tray.tmax             = isec.distance;
  }
  return intersection;
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRACE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Convenience functions
[[maybe_unused]] static vec3f eval_position(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_position(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static vec3f eval_normal(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_normal(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static vec3f eval_element_normal(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_element_normal(
      scene, scene.instances[intersection.instance], intersection.element);
}
[[maybe_unused]] static vec3f eval_shading_position(const scene_data& scene,
    const scene_intersection& intersection, const vec3f& outgoing) {
  return eval_shading_position(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv, outgoing);
}
[[maybe_unused]] static vec3f eval_shading_normal(const scene_data& scene,
    const scene_intersection& intersection, const vec3f& outgoing) {
  return eval_shading_normal(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv, outgoing);
}
[[maybe_unused]] static vec2f eval_texcoord(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_texcoord(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static material_point eval_material(
    const scene_data& scene, const scene_intersection& intersection) {
  return eval_material(scene, scene.instances[intersection.instance],
      intersection.element, intersection.uv);
}
[[maybe_unused]] static bool is_volumetric(
    const scene_data& scene, const scene_intersection& intersection) {
  return is_volumetric(scene, scene.instances[intersection.instance]);
}

}  // namespace yocto

// -----------------------------------------------------------------------------
// TRACE FUNCTIONS
// -----------------------------------------------------------------------------
namespace yocto {

// Evaluates/sample the BRDF scaled by the cosine of the incoming direction.
static vec3f eval_emission(const material_point& material, const vec3f& normal,
    const vec3f& outgoing) {
  return dot(normal, outgoing) >= 0 ? material.emission : vec3f{0, 0, 0};
}

// Evaluates/sample the BRDF scaled by the cosine of the incoming direction.
static vec3f eval_bsdfcos(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness == 0) return {0, 0, 0};

  if (material.type == material_type::matte) {
    return eval_matte(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::glossy) {
    return eval_glossy(material.color, material.ior, material.roughness, normal,
        outgoing, incoming);
  } else if (material.type == material_type::reflective) {
    return eval_reflective(
        material.color, material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return eval_transparent(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return eval_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::subsurface) {
    return eval_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::gltfpbr) {
    return eval_gltfpbr(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, incoming);
  } else {
    return {0, 0, 0};
  }
}

static vec3f eval_delta(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness != 0) return {0, 0, 0};

  if (material.type == material_type::reflective) {
    return eval_reflective(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return eval_transparent(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return eval_refractive(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::volumetric) {
    return eval_passthrough(material.color, normal, outgoing, incoming);
  } else {
    return {0, 0, 0};
  }
}

// Picks a direction based on the BRDF
static vec3f sample_bsdfcos(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, float rnl, const vec2f& rn) {
  if (material.roughness == 0) return {0, 0, 0};

  if (material.type == material_type::matte) {
    return sample_matte(material.color, normal, outgoing, rn);
  } else if (material.type == material_type::glossy) {
    return sample_glossy(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::reflective) {
    return sample_reflective(
        material.color, material.roughness, normal, outgoing, rn);
  } else if (material.type == material_type::transparent) {
    return sample_transparent(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::refractive) {
    return sample_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::subsurface) {
    return sample_refractive(material.color, material.ior, material.roughness,
        normal, outgoing, rnl, rn);
  } else if (material.type == material_type::gltfpbr) {
    return sample_gltfpbr(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, rnl, rn);
  } else {
    return {0, 0, 0};
  }
}

static vec3f sample_delta(const material_point& material, const vec3f& normal,
    const vec3f& outgoing, float rnl) {
  if (material.roughness != 0) return {0, 0, 0};

  if (material.type == material_type::reflective) {
    return sample_reflective(material.color, normal, outgoing);
  } else if (material.type == material_type::transparent) {
    return sample_transparent(
        material.color, material.ior, normal, outgoing, rnl);
  } else if (material.type == material_type::refractive) {
    return sample_refractive(
        material.color, material.ior, normal, outgoing, rnl);
  } else if (material.type == material_type::volumetric) {
    return sample_passthrough(material.color, normal, outgoing);
  } else {
    return {0, 0, 0};
  }
}

// Compute the weight for sampling the BRDF
static float sample_bsdfcos_pdf(const material_point& material,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness == 0) return 0;

  if (material.type == material_type::matte) {
    return sample_matte_pdf(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::glossy) {
    return sample_glossy_pdf(material.color, material.ior, material.roughness,
        normal, outgoing, incoming);
  } else if (material.type == material_type::reflective) {
    return sample_reflective_pdf(
        material.color, material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return sample_tranparent_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return sample_refractive_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::subsurface) {
    return sample_refractive_pdf(material.color, material.ior,
        material.roughness, normal, outgoing, incoming);
  } else if (material.type == material_type::gltfpbr) {
    return sample_gltfpbr_pdf(material.color, material.ior, material.roughness,
        material.metallic, normal, outgoing, incoming);
  } else {
    return 0;
  }
}

static float sample_delta_pdf(const material_point& material,
    const vec3f& normal, const vec3f& outgoing, const vec3f& incoming) {
  if (material.roughness != 0) return 0;

  if (material.type == material_type::reflective) {
    return sample_reflective_pdf(material.color, normal, outgoing, incoming);
  } else if (material.type == material_type::transparent) {
    return sample_tranparent_pdf(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::refractive) {
    return sample_refractive_pdf(
        material.color, material.ior, normal, outgoing, incoming);
  } else if (material.type == material_type::volumetric) {
    return sample_passthrough_pdf(material.color, normal, outgoing, incoming);
  } else {
    return 0;
  }
}

static vec3f eval_scattering(const material_point& material,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.density == vec3f{0, 0, 0}) return {0, 0, 0};
  return material.scattering * material.density *
         eval_phasefunction(material.scanisotropy, outgoing, incoming);
}

static vec3f sample_scattering(const material_point& material,
    const vec3f& outgoing, float rnl, const vec2f& rn) {
  if (material.density == vec3f{0, 0, 0}) return {0, 0, 0};
  return sample_phasefunction(material.scanisotropy, outgoing, rn);
}

static float sample_scattering_pdf(const material_point& material,
    const vec3f& outgoing, const vec3f& incoming) {
  if (material.density == vec3f{0, 0, 0}) return 0;
  return sample_phasefunction_pdf(material.scanisotropy, outgoing, incoming);
}

// Sample camera
static ray3f sample_camera(const camera_data& camera, const vec2i& ij,
    const vec2i& image_size, const vec2f& puv, const vec2f& luv, bool tent) {
  if (!tent) {
    auto uv = vec2f{
        (ij.x + puv.x) / image_size.x, (ij.y + puv.y) / image_size.y};
    return eval_camera(camera, uv, sample_disk(luv));
  } else {
    const auto width  = 2.0f;
    const auto offset = 0.5f;
    auto       fuv =
        width *
            vec2f{
                puv.x < 0.5f ? sqrt(2 * puv.x) - 1 : 1 - sqrt(2 - 2 * puv.x),
                puv.y < 0.5f ? sqrt(2 * puv.y) - 1 : 1 - sqrt(2 - 2 * puv.y),
            } +
        offset;
    auto uv = vec2f{
        (ij.x + fuv.x) / image_size.x, (ij.y + fuv.y) / image_size.y};
    return eval_camera(camera, uv, sample_disk(luv));
  }
}

// Sample lights wrt solid angle
static vec3f sample_lights(const scene_data& scene, const trace_lights& lights,
    const vec3f& position, float rl, float rel, const vec2f& ruv) {
  auto  light_id = sample_uniform((int)lights.lights.size(), rl);
  auto& light    = lights.lights[light_id];
  if (light.instance != invalidid) {
    auto& instance  = scene.instances[light.instance];
    auto& shape     = scene.shapes[instance.shape];
    auto  element   = sample_discrete(light.elements_cdf, rel);
    auto  uv        = (!shape.triangles.empty()) ? sample_triangle(ruv) : ruv;
    auto  lposition = eval_position(scene, instance, element, uv);
    return normalize(lposition - position);
  } else if (light.environment != invalidid) {
    auto& environment = scene.environments[light.environment];
    if (environment.emission_tex != invalidid) {
      auto& emission_tex = scene.textures[environment.emission_tex];
      auto  idx          = sample_discrete(light.elements_cdf, rel);
      auto  uv = vec2f{((idx % emission_tex.width) + 0.5f) / emission_tex.width,
          ((idx / emission_tex.width) + 0.5f) / emission_tex.height};
      return transform_direction(environment.frame,
          {cos(uv.x * 2 * pif) * sin(uv.y * pif), cos(uv.y * pif),
              sin(uv.x * 2 * pif) * sin(uv.y * pif)});
    } else {
      return sample_sphere(ruv);
    }
  } else {
    return {0, 0, 0};
  }
}

// Sample lights pdf
static float sample_lights_pdf(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const vec3f& position, const vec3f& direction) {
  auto pdf = 0.0f;
  for (auto& light : lights.lights) {
    if (light.instance != invalidid) {
      auto& instance = scene.instances[light.instance];
      // check all intersection
      auto lpdf          = 0.0f;
      auto next_position = position;
      for (auto bounce = 0; bounce < 100; bounce++) {
        auto intersection = intersect_instance(
            bvh, scene, light.instance, {next_position, direction});
        if (!intersection.hit) break;
        // accumulate pdf
        auto lposition = eval_position(
            scene, instance, intersection.element, intersection.uv);
        auto lnormal = eval_element_normal(
            scene, instance, intersection.element);
        // prob triangle * area triangle = area triangle mesh
        auto area = light.elements_cdf.back();
        lpdf += distance_squared(lposition, position) /
                (abs(dot(lnormal, direction)) * area);
        // continue
        next_position = lposition + direction * 1e-3f;
      }
      pdf += lpdf;
    } else if (light.environment != invalidid) {
      auto& environment = scene.environments[light.environment];
      if (environment.emission_tex != invalidid) {
        auto& emission_tex = scene.textures[environment.emission_tex];
        auto  wl = transform_direction_inverse(environment.frame, direction);
        auto  texcoord = vec2f{atan2(wl.z, wl.x) / (2 * pif),
            acos(clamp(wl.y, -1.0f, 1.0f)) / pif};
        if (texcoord.x < 0) texcoord.x += 1;
        auto i = clamp(
            (int)(texcoord.x * emission_tex.width), 0, emission_tex.width - 1);
        auto j    = clamp((int)(texcoord.y * emission_tex.height), 0,
               emission_tex.height - 1);
        auto prob = sample_discrete_pdf(
                        light.elements_cdf, j * emission_tex.width + i) /
                    light.elements_cdf.back();
        auto angle = (2 * pif / emission_tex.width) *
                     (pif / emission_tex.height) *
                     sin(pif * (j + 0.5f) / emission_tex.height);
        pdf += prob / angle;
      } else {
        pdf += 1 / (4 * pif);
      }
    }
  }
  pdf *= sample_uniform_pdf((int)lights.lights.size());
  return pdf;
}

struct trace_result {
  vec3f radiance = {0, 0, 0};
  bool  hit      = false;
  vec3f albedo   = {0, 0, 0};
  vec3f normal   = {0, 0, 0};
};

// Recursive path tracing.
static trace_result trace_path(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto opbounce      = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
           vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      radiance += weight * eval_emission(material, normal, outgoing);

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        if (rand1f(rng) < 0.5f) {
          incoming = sample_bsdfcos(
              material, normal, outgoing, rand1f(rng), rand2f(rng));
        } else {
          incoming = sample_lights(
              scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        }
        if (incoming == vec3f{0, 0, 0}) break;
        weight *=
            eval_bsdfcos(material, normal, outgoing, incoming) /
            (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
                0.5f *
                    sample_lights_pdf(scene, bvh, lights, position, incoming));
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // accumulate emission
      // radiance += weight * eval_volemission(emission, outgoing);

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) break;
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing.
static trace_result trace_pathdirect(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray_,
    rng_state& rng, const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto next_emission = true;
  auto opbounce      = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if ((bounce > 0 || !params.envhidden) && next_emission)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
           vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      if (next_emission)
        radiance += weight * eval_emission(material, normal, outgoing);

      // direct
      if (!is_delta(material)) {
        auto incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        auto pdf = sample_lights_pdf(scene, bvh, lights, position, incoming);
        auto bsdfcos = eval_bsdfcos(material, normal, outgoing, incoming);
        if (bsdfcos != vec3f{0, 0, 0} && pdf > 0) {
          auto intersection = intersect_scene(bvh, scene, {position, incoming});
          auto emission =
              !intersection.hit
                  ? eval_environment(scene, incoming)
                  : eval_emission(eval_material(scene,
                                      scene.instances[intersection.instance],
                                      intersection.element, intersection.uv),
                        eval_shading_normal(scene,
                            scene.instances[intersection.instance],
                            intersection.element, intersection.uv, -incoming),
                        -incoming);
          radiance += weight * bsdfcos * emission / pdf;
        }
        next_emission = false;
      } else {
        next_emission = true;
      }

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        if (rand1f(rng) < 0.5f) {
          incoming = sample_bsdfcos(
              material, normal, outgoing, rand1f(rng), rand2f(rng));
        } else {
          incoming = sample_lights(
              scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        }
        if (incoming == vec3f{0, 0, 0}) break;
        weight *=
            eval_bsdfcos(material, normal, outgoing, incoming) /
            (0.5f * sample_bsdfcos_pdf(material, normal, outgoing, incoming) +
                0.5f *
                    sample_lights_pdf(scene, bvh, lights, position, incoming));
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        if (incoming == vec3f{0, 0, 0}) break;
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
      }
      if (incoming == vec3f{0, 0, 0}) break;
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing with MIS.
static trace_result trace_pathmis(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance      = vec3f{0, 0, 0};
  auto weight        = vec3f{1, 1, 1};
  auto ray           = ray_;
  auto volume_stack  = svector<material_point>{};
  auto max_roughness = 0.0f;
  auto hit           = false;
  auto hit_albedo    = vec3f{0, 0, 0};
  auto hit_normal    = vec3f{0, 0, 0};
  auto opbounce      = 0;

  // MIS helpers
  auto mis_heuristic = [](float this_pdf, float other_pdf) {
    return (this_pdf * this_pdf) /
           (this_pdf * this_pdf + other_pdf * other_pdf);
  };
  auto next_emission     = true;
  auto next_intersection = scene_intersection{};

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = next_emission ? intersect_scene(bvh, scene, ray)
                                      : next_intersection;
    if (!intersection.hit) {
      if ((bounce > 0 || !params.envhidden) && next_emission)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // handle transmission if inside a volume
    auto in_volume = false;
    if (!volume_stack.empty()) {
      auto& vsdf     = volume_stack.back();
      auto  distance = sample_transmittance(
           vsdf.density, intersection.distance, rand1f(rng), rand1f(rng));
      weight *= eval_transmittance(vsdf.density, distance) /
                sample_transmittance_pdf(
                    vsdf.density, distance, intersection.distance);
      in_volume             = distance < intersection.distance;
      intersection.distance = distance;
    }

    // switch between surface and volume
    if (!in_volume) {
      // prepare shading point
      auto outgoing = -ray.d;
      auto position = eval_shading_position(scene, intersection, outgoing);
      auto normal   = eval_shading_normal(scene, intersection, outgoing);
      auto material = eval_material(scene, intersection);

      // correct roughness
      if (params.nocaustics) {
        max_roughness      = max(material.roughness, max_roughness);
        material.roughness = max_roughness;
      }

      // handle opacity
      if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
        if (opbounce++ > 128) break;
        ray = {position + ray.d * 1e-2f, ray.d};
        bounce -= 1;
        continue;
      }

      // set hit variables
      if (bounce == 0) {
        hit        = true;
        hit_albedo = material.color;
        hit_normal = normal;
      }

      // accumulate emission
      if (next_emission) {
        radiance += weight * eval_emission(material, normal, outgoing);
      }

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (!is_delta(material)) {
        // direct with MIS --- light
        for (auto sample_light : {true, false}) {
          incoming = sample_light ? sample_lights(scene, lights, position,
                                        rand1f(rng), rand1f(rng), rand2f(rng))
                                  : sample_bsdfcos(material, normal, outgoing,
                                        rand1f(rng), rand2f(rng));
          if (incoming == vec3f{0, 0, 0}) break;
          auto bsdfcos   = eval_bsdfcos(material, normal, outgoing, incoming);
          auto light_pdf = sample_lights_pdf(
              scene, bvh, lights, position, incoming);
          auto bsdf_pdf = sample_bsdfcos_pdf(
              material, normal, outgoing, incoming);
          auto mis_weight = sample_light
                                ? mis_heuristic(light_pdf, bsdf_pdf) / light_pdf
                                : mis_heuristic(bsdf_pdf, light_pdf) / bsdf_pdf;
          if (bsdfcos != vec3f{0, 0, 0} && mis_weight != 0) {
            auto intersection = intersect_scene(
                bvh, scene, {position, incoming});
            if (!sample_light) next_intersection = intersection;
            auto emission = vec3f{0, 0, 0};
            if (!intersection.hit) {
              emission = eval_environment(scene, incoming);
            } else {
              auto material = eval_material(scene,
                  scene.instances[intersection.instance], intersection.element,
                  intersection.uv);
              emission      = eval_emission(material,
                       eval_shading_normal(scene,
                           scene.instances[intersection.instance],
                           intersection.element, intersection.uv, -incoming),
                       -incoming);
            }
            radiance += weight * bsdfcos * emission * mis_weight;
          }
        }

        // indirect
        weight *= eval_bsdfcos(material, normal, outgoing, incoming) /
                  sample_bsdfcos_pdf(material, normal, outgoing, incoming);
        next_emission = false;
      } else {
        incoming = sample_delta(material, normal, outgoing, rand1f(rng));
        weight *= eval_delta(material, normal, outgoing, incoming) /
                  sample_delta_pdf(material, normal, outgoing, incoming);
        next_emission = true;
      }

      // update volume stack
      if (is_volumetric(scene, intersection) &&
          dot(normal, outgoing) * dot(normal, incoming) < 0) {
        if (volume_stack.empty()) {
          auto material = eval_material(scene, intersection);
          volume_stack.push_back(material);
        } else {
          volume_stack.pop_back();
        }
      }

      // setup next iteration
      ray = {position, incoming};
    } else {
      // prepare shading point
      auto  outgoing = -ray.d;
      auto  position = ray.o + ray.d * intersection.distance;
      auto& vsdf     = volume_stack.back();

      // next direction
      auto incoming = vec3f{0, 0, 0};
      if (rand1f(rng) < 0.5f) {
        incoming = sample_scattering(vsdf, outgoing, rand1f(rng), rand2f(rng));
        next_emission = true;
      } else {
        incoming = sample_lights(
            scene, lights, position, rand1f(rng), rand1f(rng), rand2f(rng));
        next_emission = true;
      }
      weight *=
          eval_scattering(vsdf, outgoing, incoming) /
          (0.5f * sample_scattering_pdf(vsdf, outgoing, incoming) +
              0.5f * sample_lights_pdf(scene, bvh, lights, position, incoming));

      // setup next iteration
      ray = {position, incoming};
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Recursive path tracing.
static trace_result trace_naive(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    radiance += weight * eval_emission(material, normal, outgoing);

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (material.roughness != 0) {
      incoming = sample_bsdfcos(
          material, normal, outgoing, rand1f(rng), rand2f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_bsdfcos(material, normal, outgoing, incoming) /
                sample_bsdfcos_pdf(material, normal, outgoing, incoming);
    } else {
      incoming = sample_delta(material, normal, outgoing, rand1f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_delta(material, normal, outgoing, incoming) /
                sample_delta_pdf(material, normal, outgoing, incoming);
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }

    // setup next iteration
    ray = {position, incoming};
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Eyelight for quick previewing.
static trace_result trace_eyelight(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray_,
    rng_state& rng, const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;

  // trace  path
  for (auto bounce = 0; bounce < max(params.bounces, 4); bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    auto incoming = outgoing;
    radiance += weight * eval_emission(material, normal, outgoing);

    // brdf * light
    radiance += weight * pif *
                eval_bsdfcos(material, normal, outgoing, incoming);

    // continue path
    if (!is_delta(material)) break;
    incoming = sample_delta(material, normal, outgoing, rand1f(rng));
    if (incoming == vec3f{0, 0, 0}) break;
    weight *= eval_delta(material, normal, outgoing, incoming) /
              sample_delta_pdf(material, normal, outgoing, incoming);
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // setup next iteration
    ray = {position, incoming};
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Eyelight with ambient occlusion for quick previewing.
static trace_result trace_eyelightao(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray_,
    rng_state& rng, const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;

  // trace  path
  for (auto bounce = 0; bounce < max(params.bounces, 4); bounce++) {
    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto outgoing = -ray.d;
    auto position = eval_shading_position(scene, intersection, outgoing);
    auto normal   = eval_shading_normal(scene, intersection, outgoing);
    auto material = eval_material(scene, intersection);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    auto incoming = outgoing;
    radiance += weight * eval_emission(material, normal, outgoing);

    // occlusion
    auto occluding = sample_hemisphere_cos(normal, rand2f(rng));
    if (intersect_scene(bvh, scene, {position, occluding}).hit) break;

    // brdf * light
    radiance += weight * pif *
                eval_bsdfcos(material, normal, outgoing, incoming);

    // continue path
    if (!is_delta(material)) break;
    incoming = sample_delta(material, normal, outgoing, rand1f(rng));
    if (incoming == vec3f{0, 0, 0}) break;
    weight *= eval_delta(material, normal, outgoing, incoming) /
              sample_delta_pdf(material, normal, outgoing, incoming);
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // setup next iteration
    ray = {position, incoming};
  }

  return {radiance, hit, hit_albedo, hit_normal};
}

// Furnace test.
static trace_result trace_furnace(const scene_data& scene, const trace_bvh& bvh,
    const trace_lights& lights, const ray3f& ray_, rng_state& rng,
    const trace_params& params) {
  // initialize
  auto radiance   = vec3f{0, 0, 0};
  auto weight     = vec3f{1, 1, 1};
  auto ray        = ray_;
  auto hit        = false;
  auto hit_albedo = vec3f{0, 0, 0};
  auto hit_normal = vec3f{0, 0, 0};
  auto opbounce   = 0;
  auto in_volume  = false;

  // trace  path
  for (auto bounce = 0; bounce < params.bounces; bounce++) {
    // exit loop
    if (bounce > 0 && !in_volume) {
      radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // intersect next point
    auto intersection = intersect_scene(bvh, scene, ray);
    if (!intersection.hit) {
      if (bounce > 0 || !params.envhidden)
        radiance += weight * eval_environment(scene, ray.d);
      break;
    }

    // prepare shading point
    auto  outgoing = -ray.d;
    auto& instance = scene.instances[intersection.instance];
    auto  element  = intersection.element;
    auto  uv       = intersection.uv;
    auto  position = eval_position(scene, instance, element, uv);
    auto  normal = eval_shading_normal(scene, instance, element, uv, outgoing);
    auto  material = eval_material(scene, instance, element, uv);

    // handle opacity
    if (material.opacity < 1 && rand1f(rng) >= material.opacity) {
      if (opbounce++ > 128) break;
      ray = {position + ray.d * 1e-2f, ray.d};
      bounce -= 1;
      continue;
    }

    // set hit variables
    if (bounce == 0) {
      hit        = true;
      hit_albedo = material.color;
      hit_normal = normal;
    }

    // accumulate emission
    radiance += weight * eval_emission(material, normal, outgoing);

    // next direction
    auto incoming = vec3f{0, 0, 0};
    if (material.roughness != 0) {
      incoming = sample_bsdfcos(
          material, normal, outgoing, rand1f(rng), rand2f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_bsdfcos(material, normal, outgoing, incoming) /
                sample_bsdfcos_pdf(material, normal, outgoing, incoming);
    } else {
      incoming = sample_delta(material, normal, outgoing, rand1f(rng));
      if (incoming == vec3f{0, 0, 0}) break;
      weight *= eval_delta(material, normal, outgoing, incoming) /
                sample_delta_pdf(material, normal, outgoing, incoming);
    }

    // check weight
    if (weight == vec3f{0, 0, 0} || !isfinite(weight)) break;

    // russian roulette
    if (bounce > 3) {
      auto rr_prob = min((float)0.99, max(weight));
      if (rand1f(rng) >= rr_prob) break;
      weight *= 1 / rr_prob;
    }

    // update volume stack
    if (dot(normal, outgoing) * dot(normal, incoming) < 0)
      in_volume = !in_volume;

    // setup next iteration
    ray = {position, incoming};
  }

  // done
  return {radiance, hit, hit_albedo, hit_normal};
}

// False color rendering
static trace_result trace_falsecolor(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray,
    rng_state& rng, const trace_params& params) {
  // intersect next point
  auto intersection = intersect_scene(bvh, scene, ray);
  if (!intersection.hit) return {};

  // prepare shading point
  auto outgoing = -ray.d;
  auto position = eval_shading_position(scene, intersection, outgoing);
  auto normal   = eval_shading_normal(scene, intersection, outgoing);
  auto gnormal  = eval_element_normal(scene, intersection);
  auto texcoord = eval_texcoord(scene, intersection);
  auto material = eval_material(scene, intersection);
  auto delta    = is_delta(material) ? 1.0f : 0.0f;

  // hash color
  auto hashed_color = [](int id) {
    auto rng = make_rng(trace_default_seed, id * 2 + 1);
    return pow(0.5f + 0.5f * rand3f(rng), 2.2f);
  };

  // compute result
  auto result = vec3f{0, 0, 0};
  switch (params.falsecolor) {
    case trace_falsecolor_type::position:
      result = position * 0.5f + 0.5f;
      break;
    case trace_falsecolor_type::normal: result = normal * 0.5f + 0.5f; break;
    case trace_falsecolor_type::frontfacing:
      result = dot(normal, -ray.d) > 0 ? vec3f{0, 1, 0} : vec3f{1, 0, 0};
      break;
    case trace_falsecolor_type::gnormal: result = gnormal * 0.5f + 0.5f; break;
    case trace_falsecolor_type::gfrontfacing:
      result = dot(gnormal, -ray.d) > 0 ? vec3f{0, 1, 0} : vec3f{1, 0, 0};
      break;
    case trace_falsecolor_type::mtype:
      result = hashed_color((int)material.type);
      break;
    case trace_falsecolor_type::texcoord:
      result = {fmod(texcoord.x, 1.0f), fmod(texcoord.y, 1.0f), 0};
      break;
    case trace_falsecolor_type::color: result = material.color; break;
    case trace_falsecolor_type::emission: result = material.emission; break;
    case trace_falsecolor_type::roughness:
      result = {material.roughness, material.roughness, material.roughness};
      break;
    case trace_falsecolor_type::opacity:
      result = {material.opacity, material.opacity, material.opacity};
      break;
    case trace_falsecolor_type::metallic:
      result = {material.metallic, material.metallic, material.metallic};
      break;
    case trace_falsecolor_type::delta: result = {delta, delta, delta}; break;
    case trace_falsecolor_type::element:
      result = hashed_color(intersection.element);
      break;
    case trace_falsecolor_type::instance:
      result = hashed_color(intersection.instance);
      break;
    case trace_falsecolor_type::shape:
      result = hashed_color(scene.instances[intersection.instance].shape);
      break;
    case trace_falsecolor_type::material:
      result = hashed_color(scene.instances[intersection.instance].material);
      break;
    case trace_falsecolor_type::highlight: {
      if (material.emission == vec3f{0, 0, 0})
        material.emission = {0.2f, 0.2f, 0.2f};
      result = material.emission * abs(dot(-ray.d, normal));
      break;
    } break;
    default: result = {0, 0, 0};
  }

  // done
  return {srgb_to_rgb(result), true, material.color, normal};
}

// Trace a single ray from the camera using the given algorithm.
using sampler_func = trace_result (*)(const scene_data& scene,
    const trace_bvh& bvh, const trace_lights& lights, const ray3f& ray,
    rng_state& rng, const trace_params& params);
static sampler_func get_trace_sampler_func(const trace_params& params) {
  switch (params.sampler) {
    case trace_sampler_type::path: return trace_path;
    case trace_sampler_type::pathdirect: return trace_pathdirect;
    case trace_sampler_type::pathmis: return trace_pathmis;
    case trace_sampler_type::naive: return trace_naive;
    case trace_sampler_type::eyelight: return trace_eyelight;
    case trace_sampler_type::eyelightao: return trace_eyelightao;
    case trace_sampler_type::furnace: return trace_furnace;
    case trace_sampler_type::falsecolor: return trace_falsecolor;
    default: {
      return nullptr;
    }
  }
}

static void trace_sample(cutrace_state& state, const cutrace_scene& scene,
    const cutrace_bvh& bvh, const cutrace_lights& lights, int i, int j,
    const cutrace_params& params) {
  auto& camera  = scene.cameras[params.camera];
  auto  sampler = get_trace_sampler_func(params);
  auto  idx     = state.width * j + i;
  auto  ray     = sample_camera(camera, {i, j}, {state.width, state.height},
           rand2f(state.rngs[idx]), rand2f(state.rngs[idx]), params.tentfilter);
  auto  result  = sampler(scene, bvh, lights, ray, state.rngs[idx], params);
  // auto [radiance, hit, albedo, normal] = sampler(
  //    scene, bvh, lights, ray, state.rngs[idx], params);
  auto radiance = result.radiance;
  auto hit      = result.hit;
  auto albedo   = result.albedo;
  auto normal   = result.normal;
  if (!isfinite(radiance)) radiance = {0, 0, 0};
  if (max(radiance) > params.clamp)
    radiance = radiance * (params.clamp / max(radiance));
  if (hit) {
    state.image[idx] += {radiance.x, radiance.y, radiance.z, 1};
    state.albedo[idx] += albedo;
    state.normal[idx] += normal;
    state.hits[idx] += 1;
  } else if (!params.envhidden && !scene.environments.empty()) {
    state.image[idx] += {radiance.x, radiance.y, radiance.z, 1};
    state.albedo[idx] += {1, 1, 1};
    state.normal[idx] += -ray.d;
    state.hits[idx] += 1;
  }
}

// raygen shader
optix_shader void __raygen__trace_pixel() {
  // pixel index
  auto ij  = optixGetLaunchIndex();
  auto idx = ij.y * globals.state.width + ij.x;

  // initialize state on first sample
  if (globals.state.samples == 0) {
    globals.state.image[idx] = {0, 0, 0, 0};
    globals.state.rngs[idx]  = make_rng(98273987, idx * 2 + 1);
  }

  // run shading
  auto nsamples = 256;
  for (auto sample = 0; sample < nsamples; sample++) {
    trace_sample(globals.state, globals.scene, globals.bvh, globals.lights,
        optixGetLaunchIndex().x, optixGetLaunchIndex().y, cutrace_params{});
  }

  // normalize output
  globals.state.image[idx] /= nsamples;
}

}  // namespace yocto